#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <memory.h>
#include <sys/types.h> // off_t

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_helper_alexis.h"
#include "cuda_vectors.h"
#include "cuda_vector_uint2x4.h"

#define ROTR(x,n) ROTR64(x,n)

// use sp kernel on SM 5+
#define SP_KERNEL

#define USE_SHUFFLE 0

__constant__
static uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)

// ---------------------------- BEGIN CUDA quark_blake512 functions ------------------------------------

__device__ __constant__
static const uint8_t c_sigma_big[16][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },

	{12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },

	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

__device__ __constant__
static const uint64_t c_u512[16] =
{
	0x243f6a8885a308d3ULL, 0x13198a2e03707344ULL,
	0xa4093822299f31d0ULL, 0x082efa98ec4e6c89ULL,
	0x452821e638d01377ULL, 0xbe5466cf34e90c6cULL,
	0xc0ac29b7c97c50ddULL, 0x3f84d5b5b5470917ULL,
	0x9216d5d98979fb1bULL, 0xd1310ba698dfb5acULL,
	0x2ffd72dbd01adfb7ULL, 0xb8e1afed6a267e96ULL,
	0xba7c9045f12c7f99ULL, 0x24a19947b3916cf7ULL,
	0x0801f2e2858efc16ULL, 0x636920d871574e69ULL
};

#define G(a,b,c,d,x) { \
	uint32_t idx1 = sigma[i][x]; \
	uint32_t idx2 = sigma[i][x+1]; \
	v[a] += (m[idx1] ^ u512[idx2]) + v[b]; \
	v[d] = SWAPDWORDS(v[d] ^ v[a]); \
	v[c] += v[d]; \
	v[b] = ROTR( v[b] ^ v[c], 25); \
	v[a] += (m[idx2] ^ u512[idx1]) + v[b]; \
	v[d] = ROTR( v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = ROTR( v[b] ^ v[c], 11); \
}

__device__ __forceinline__
void quark_blake512_compress(uint64_t *h, const uint64_t *block, const uint8_t ((*sigma)[16]), const uint64_t *u512, const int T0)
{
	uint64_t v[16];
	uint64_t m[16];

	#pragma unroll
	for(int i=0; i < 16; i++) {
		m[i] = cuda_swab64(block[i]);
	}

	//#pragma unroll 8
	for(int i=0; i < 8; i++)
		v[i] = h[i];

	v[ 8] = u512[0];
	v[ 9] = u512[1];
	v[10] = u512[2];
	v[11] = u512[3];
	v[12] = u512[4] ^ T0;
	v[13] = u512[5] ^ T0;
	v[14] = u512[6];
	v[15] = u512[7];

	//#pragma unroll 16
	for(int i=0; i < 16; i++)
	{
		/* column step */
		G( 0, 4, 8, 12, 0 );
		G( 1, 5, 9, 13, 2 );
		G( 2, 6, 10, 14, 4 );
		G( 3, 7, 11, 15, 6 );
		/* diagonal step */
		G( 0, 5, 10, 15, 8 );
		G( 1, 6, 11, 12, 10 );
		G( 2, 7, 8, 13, 12 );
		G( 3, 4, 9, 14, 14 );
	}

	h[0] ^= v[0] ^ v[8];
	h[1] ^= v[1] ^ v[9];
	h[2] ^= v[2] ^ v[10];
	h[3] ^= v[3] ^ v[11];
	h[4] ^= v[4] ^ v[12];
	h[5] ^= v[5] ^ v[13];
	h[6] ^= v[6] ^ v[14];
	h[7] ^= v[7] ^ v[15];
}

__global__ __launch_bounds__(256, 4)
void quark_blake512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *g_nonceVector, uint64_t *g_hash)
{
#if !defined(SP_KERNEL) || __CUDA_ARCH__ < 500
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

#if USE_SHUFFLE
	const uint32_t warpBlockID = (thread + 15)>>4; // aufrunden auf volle Warp-Blöcke

	if (warpBlockID < ( (threads+15)>>4 ))
#else
	if (thread < threads)
#endif
	{
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		off_t hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[hashPosition<<3]; // hashPosition * 8

		// 128 Bytes
		uint64_t buf[16];

		// State
		uint64_t h[8] = {
			0x6a09e667f3bcc908ULL,
			0xbb67ae8584caa73bULL,
			0x3c6ef372fe94f82bULL,
			0xa54ff53a5f1d36f1ULL,
			0x510e527fade682d1ULL,
			0x9b05688c2b3e6c1fULL,
			0x1f83d9abfb41bd6bULL,
			0x5be0cd19137e2179ULL
		};

		// Message for first round
		#pragma unroll 8
		for (int i=0; i < 8; ++i)
			buf[i] = inpHash[i];

		// Hash Pad
		buf[8]  = 0x0000000000000080ull;
		buf[9]  = 0;
		buf[10] = 0;
		buf[11] = 0;
		buf[12] = 0;
		buf[13] = 0x0100000000000000ull;
		buf[14] = 0;
		buf[15] = 0x0002000000000000ull;

		// Ending round
		quark_blake512_compress(h, buf, c_sigma_big, c_u512, 512);

#if __CUDA_ARCH__ <= 350
		uint32_t *outHash = (uint32_t*)&g_hash[hashPosition * 8U];
		#pragma unroll 8
		for (int i=0; i < 8; i++) {
			outHash[2*i+0] = cuda_swab32( _HIDWORD(h[i]) );
			outHash[2*i+1] = cuda_swab32( _LODWORD(h[i]) );
		}
#else
		uint64_t *outHash = &g_hash[hashPosition * 8U];
		for (int i=0; i < 8; i++) {
			outHash[i] = cuda_swab64(h[i]);
		}
#endif
	}
#endif /* SP */
}

__global__ __launch_bounds__(256,4)
void quark_blake512_gpu_hash_80(uint32_t threads, uint32_t startNounce, void *outputHash)
{
//#if !defined(SP_KERNEL) || __CUDA_ARCH__ < 500
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t buf[16];
		#pragma unroll
		for (int i=0; i < 16; ++i)
			buf[i] = c_PaddedMessage80[i];

		// The test Nonce
		const uint32_t nounce = startNounce + thread;
		((uint32_t*)buf)[19] = cuda_swab32(nounce);

		uint64_t h[8] = {
			0x6a09e667f3bcc908ULL,
			0xbb67ae8584caa73bULL,
			0x3c6ef372fe94f82bULL,
			0xa54ff53a5f1d36f1ULL,
			0x510e527fade682d1ULL,
			0x9b05688c2b3e6c1fULL,
			0x1f83d9abfb41bd6bULL,
			0x5be0cd19137e2179ULL
		};

		quark_blake512_compress(h, buf, c_sigma_big, c_u512, 640);

#if __CUDA_ARCH__ <= 350
		uint32_t *outHash = (uint32_t*)outputHash + (thread * 16U);
		#pragma unroll 8
		for (uint32_t i=0; i < 8; i++) {
			outHash[2*i]   = cuda_swab32( _HIDWORD(h[i]) );
			outHash[2*i+1] = cuda_swab32( _LODWORD(h[i]) );
		}
#else
		uint64_t *outHash = (uint64_t*)outputHash + (thread * 8U);
		for (uint32_t i=0; i < 8; i++) {
			outHash[i] = cuda_swab64( h[i] );
		}
#endif
	}
//#endif
}

#ifdef SP_KERNEL
#include "cuda_quark_blake512_sp.cuh"
#endif

__host__
void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_outputHash, int order)
{
#ifdef SP_KERNEL
	int dev_id = device_map[thr_id];
	if (device_sm[dev_id] >= 500 && cuda_arch[dev_id] >= 500)
		quark_blake512_cpu_hash_64_sp(threads, startNounce, d_nonceVector, d_outputHash);
	else
#endif
	{
		const uint32_t threadsperblock = 256;
		dim3 grid((threads + threadsperblock-1)/threadsperblock);
		dim3 block(threadsperblock);
		quark_blake512_gpu_hash_64<<<grid, block>>>(threads, startNounce, d_nonceVector, (uint64_t*)d_outputHash);
	}
	//MyStreamSynchronize(NULL, order, thr_id);
}

__host__
void quark_blake512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash)
{
#ifdef SP_KERNEL
	int dev_id = device_map[thr_id];
	if (device_sm[dev_id] >= 500 && cuda_arch[dev_id] >= 500)
		quark_blake512_cpu_hash_80_sp(threads, startNounce, d_outputHash);
	else
#endif
	{
		const uint32_t threadsperblock = 256;
		dim3 grid((threads + threadsperblock-1)/threadsperblock);
		dim3 block(threadsperblock);

		quark_blake512_gpu_hash_80<<<grid, block>>>(threads, startNounce, d_outputHash);
	}
}

// ---------------------------- END CUDA quark_blake512 functions ------------------------------------

__host__
void quark_blake512_cpu_init(int thr_id, uint32_t threads)
{
	cuda_get_arch(thr_id);
}

__host__
void quark_blake512_cpu_free(int thr_id)
{
}

// ----------------------------- Host midstate for 80-bytes input ------------------------------------

#undef SPH_C32
#undef SPH_T32
#undef SPH_C64
#undef SPH_T64

extern "C" {
#include "sph/sph_blake.h"
}

__host__
void quark_blake512_cpu_setBlock_80(int thr_id, uint32_t *endiandata)
{
#ifdef SP_KERNEL
	int dev_id = device_map[thr_id];
	if (device_sm[dev_id] >= 500 && cuda_arch[dev_id] >= 500)
		quark_blake512_cpu_setBlock_80_sp(thr_id, (uint64_t*) endiandata);
	else
#endif
	{
		uint64_t message[16];

		memcpy(message, endiandata, 80);
		message[10] = 0x80;
		message[11] = 0;
		message[12] = 0;
		message[13] = 0x0100000000000000ull;
		message[14] = 0;
		message[15] = 0x8002000000000000ull; // 0x280

		hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), message, sizeof(message), 0, hipMemcpyHostToDevice);
	}
	CUDA_LOG_ERROR();
}

//=====================================================================================
__constant__ uint2 _ALIGN(16) c_m[16]; // padded message (80 bytes + padding)
__constant__ uint2 _ALIGN(16) c_v[16]; //state
__constant__ uint2 _ALIGN(16) c_x[128]; //precomputed xors

#define GShost(a,b,c,d,e,f) { \
	v[a] += (m[e] ^ z[f]) + v[b]; \
	v[d] = ROTR64(v[d] ^ v[a],32); \
	v[c] += v[d]; \
	v[b] = ROTR64( v[b] ^ v[c], 25); \
	v[a] += (m[f] ^ z[e]) + v[b]; \
	v[d] = ROTR64( v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = ROTR64( v[b] ^ v[c], 11); \
}

__host__
void xevan_blake512_cpu_setBlock_80(int thr_id, uint32_t *endiandata){
	uint64_t m[16],v[16],xors[128];
	memcpy(m, endiandata, 80);
	m[10] = 0x8000000000000000ull;
	m[11] = 0;
	m[12] = 0;
	m[13] = 0x01;
	m[14] = 0;
	m[15] = 0x280;

	for(int i=0;i<10;i++){
		m[ i] = cuda_swab64(m[ i]);
	}
	
	uint64_t h[8] = {
		0x6a09e667f3bcc908ULL,	0xbb67ae8584caa73bULL,	0x3c6ef372fe94f82bULL,	0xa54ff53a5f1d36f1ULL,
		0x510e527fade682d1ULL,	0x9b05688c2b3e6c1fULL,	0x1f83d9abfb41bd6bULL,	0x5be0cd19137e2179ULL
	};

	const uint64_t z[16] = {
		0x243f6a8885a308d3ULL, 0x13198a2e03707344ULL,	0xa4093822299f31d0ULL, 0x082efa98ec4e6c89ULL,
		0x452821e638d01377ULL, 0xbe5466cf34e90c6cULL,	0xc0ac29b7c97c50ddULL, 0x3f84d5b5b5470917ULL,
		0x9216d5d98979fb1bULL, 0xd1310ba698dfb5acULL,	0x2ffd72dbd01adfb7ULL, 0xb8e1afed6a267e96ULL,
		0xba7c9045f12c7f99ULL, 0x24a19947b3916cf7ULL,	0x0801f2e2858efc16ULL, 0x636920d871574e69ULL
	};

	for(int i=0;i<8;i++){
		v[ i] = h[ i];
	}
	v[ 8] = z[0];
	v[ 9] = z[1];
	v[10] = z[2];
	v[11] = z[3];
	v[12] = z[4] ^ 640;
	v[13] = z[5] ^ 640;
	v[14] = z[6];
	v[15] = z[7];
	
	/* column step */
	GShost( 0, 4, 8,12, 0, 1);
	GShost( 1, 5, 9,13, 2, 3);
	GShost( 2, 6,10,14, 4, 5);
	GShost( 3, 7,11,15, 6, 7);

	GShost( 1, 6,11,12,10,11);
	GShost( 2, 7, 8,13,12,13);
	GShost( 3, 4, 9,14,14,15);

	v[ 0]+= (m[ 8] ^ z[ 9]) + v[ 5];
	v[15] = ROTR64(v[15]^v[ 0],32);
	v[10]+= v[15];
	v[ 5] = ROTR64(v[ 5] ^ v[10], 25);

	v[ 0]+= v[ 5];
	
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_m), m, sizeof(m), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_v), v, sizeof(m), 0, hipMemcpyHostToDevice));

	int i=0;
	
	xors[i++] = m[ 4]^z[ 8];
	xors[i++] = m[ 8]^z[ 4];	
	xors[i++] = m[ 6]^z[13];
	xors[i++] = m[ 1]^z[12];	
	xors[i++] = m[ 0]^z[ 2];
	xors[i++] = m[ 5]^z[ 3];
	xors[i++] = m[ 2]^z[ 0];	
	xors[i++] = m[ 7]^z[11];	
	xors[i++] = m[ 3]^z[ 5];
//2
	xors[i++] = m[ 5]^z[ 2];
	xors[i++] = m[ 8]^z[11];	
	xors[i++] = m[ 0]^z[12];
	xors[i++] = m[ 2]^z[ 5];
	xors[i++] = m[ 3]^z[ 6];	
	xors[i++] = m[ 7]^z[ 1];	
	xors[i++] = m[ 6]^z[ 3];	
	xors[i++] = m[ 1]^z[ 7];	
	xors[i++] = m[ 4]^z[ 9];
//3
	xors[i++] = m[ 7]^z[ 9];
	xors[i++] = m[ 3]^z[ 1];
	xors[i++] = m[ 1]^z[ 3];
	xors[i++] = m[14]^z[11];
	xors[i++] = m[ 2]^z[ 6];
	xors[i++] = m[ 5]^z[10];
	xors[i++] = m[ 4]^z[ 0];
	xors[i++] = m[ 6]^z[ 2];
	xors[i++] = m[ 0]^z[ 4];
	xors[i++] = m[ 8]^z[15];
//4
	xors[i++] = m[ 5]^z[ 7];
	xors[i++] = m[ 2]^z[ 4];
	xors[i++] = m[ 0]^z[ 9];
	xors[i++] = m[ 7]^z[ 5];
	xors[i++] = m[ 4]^z[ 2];
	xors[i++] = m[ 6]^z[ 8];
	xors[i++] = m[ 3]^z[13];
	xors[i++] = m[ 1]^z[14];
	xors[i++] = m[ 8]^z[ 6];
//5
	xors[i++] = m[ 2]^z[12];
	xors[i++] = m[ 6]^z[10];
	xors[i++] = m[ 0]^z[11];
	xors[i++] = m[ 8]^z[ 3];
	xors[i++] = m[ 3]^z[ 8];
	xors[i++] = m[ 4]^z[13];
	xors[i++] = m[ 7]^z[ 5];	
	xors[i++] = m[ 1]^z[ 9];
	xors[i++] = m[ 5]^z[ 7];
//6
	xors[i++] = m[ 1]^z[15];
	xors[i++] = m[ 4]^z[10];
	xors[i++] = m[ 5]^z[12];
	xors[i++] = m[ 0]^z[ 7];
	xors[i++] = m[ 6]^z[ 3];
	xors[i++] = m[ 8]^z[11];
	xors[i++] = m[ 7]^z[ 0];
	xors[i++] = m[ 3]^z[ 6];
	xors[i++] = m[ 2]^z[ 9];
//7
	xors[i++] = m[ 7]^z[14];
	xors[i++] = m[ 3]^z[ 9];
	xors[i++] = m[ 1]^z[12];
	xors[i++] = m[ 5]^z[ 0];
	xors[i++] = m[ 8]^z[ 6];
	xors[i++] = m[ 2]^z[10];
	xors[i++] = m[ 0]^z[ 5];
	xors[i++] = m[ 4]^z[15];
	xors[i++] = m[ 6]^z[ 8];
//8
	xors[i++] = m[ 6]^z[15];
	xors[i++] = m[ 0]^z[ 8];
	xors[i++] = m[ 3]^z[11];
	xors[i++] = m[ 8]^z[ 0];
	xors[i++] = m[ 1]^z[ 4];
	xors[i++] = m[ 2]^z[12];
	xors[i++] = m[ 7]^z[13];
	xors[i++] = m[ 4]^z[ 1];
	xors[i++] = m[ 5]^z[10];
//9
	xors[i++] = m[ 8]^z[ 4];
	xors[i++] = m[ 7]^z[ 6];
	xors[i++] = m[ 1]^z[ 5];
	xors[i++] = m[ 2]^z[10];
	xors[i++] = m[ 4]^z[ 8];
	xors[i++] = m[ 6]^z[ 7];
	xors[i++] = m[ 5]^z[ 1];
	xors[i++] = m[ 3]^z[12];
	xors[i++] = m[ 0]^z[13];
//10
	xors[i++] = m[ 0]^z[ 1];
	xors[i++] = m[ 2]^z[ 3];
	xors[i++] = m[ 4]^z[ 5];
	xors[i++] = m[ 6]^z[ 7];
	xors[i++] = m[ 1]^z[ 0];
	xors[i++] = m[ 3]^z[ 2];
	xors[i++] = m[ 5]^z[ 4];
	xors[i++] = m[ 7]^z[ 6];
	xors[i++] = m[ 8]^z[ 9];

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_x),xors, i*sizeof(uint2), 0, hipMemcpyHostToDevice));
}


//=====================================================================================

#define TPB_128_B 128

__global__ __launch_bounds__(TPB_128_B, 3)
void quark_blake512_gpu_hash_128(uint32_t threads,  uint64_t *g_hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads)
	{
		//uint64_t *inpHash = &g_hash[thread<<3]; // hashPosition * 8

		// 128 Bytes
		uint64_t buf[16];

		// State
		uint64_t h[8] = {
			0x6a09e667f3bcc908ULL,
			0xbb67ae8584caa73bULL,
			0x3c6ef372fe94f82bULL,
			0xa54ff53a5f1d36f1ULL,
			0x510e527fade682d1ULL,
			0x9b05688c2b3e6c1fULL,
			0x1f83d9abfb41bd6bULL,
			0x5be0cd19137e2179ULL
		};

		uint2x4 *phash = (uint2x4*)&g_hash[thread<<3];
		uint2x4 *outpt = (uint2x4*)buf;
		outpt[0] = __ldg4(&phash[0]);
		outpt[1] = __ldg4(&phash[1]);

		// Message for first round
		#pragma unroll 8
		for (int i=0; i < 8; ++i){
			buf[i] = cuda_swab64(buf[i]);
			//buf[i+8]=0;
		}

		// Hash Pad
		buf[8]  = 0;
		buf[9]  = 0;
		buf[10] = 0;
		buf[11] = 0;
		buf[12] = 0;
		buf[13] = 0;
		buf[14] = 0;
		buf[15] = 0;
		// Ending round
		quark_blake512_compress(h, buf, c_sigma_big, c_u512, 1024);

		buf[0] = 0x8000000000000000;
		buf[1] = 0;
		buf[2] = 0;
		buf[3] = 0;
		buf[4] = 0;
		buf[5] = 0;
		buf[6] = 0;
		buf[7] = 0;
		buf[8] = 0;
		buf[9] = 0;
		buf[10] = 0;
		buf[11] = 0;
		buf[12] = 0;
		buf[13] = 1;
		buf[14] = 0;
		buf[15] = 0x400;

		quark_blake512_compress(h, buf, c_sigma_big, c_u512, 0);

		uint64_t *outHash = &g_hash[thread * 8U];
		#pragma unroll 8
		for (int i=0; i < 8; i++) {
			outHash[i] = cuda_swab64(h[i]);
		}
	}
}

__host__
void quark_blake512_cpu_hash_128(int thr_id, uint32_t threads, uint32_t *d_outputHash)
{
		const uint32_t threadsperblock = TPB_128_B;
		dim3 grid((threads + threadsperblock-1)/threadsperblock);
		dim3 block(threadsperblock);
		quark_blake512_gpu_hash_128<<<grid, block>>>(threads,  (uint64_t*)d_outputHash);
}


//=================================================================================
#define TPB80 256

#define TPB52_64 192
#define TPB50_64 192

// Wolf's BMW512, loosely based on SPH's implementation
#define as_uint2(x) (x)
#define FAST_ROTL64_LO ROTL64
#define FAST_ROTL64_HI ROTL64

#define CONST_EXP2  q[i+0] + FAST_ROTL64_LO(as_uint2(q[i+1]), 5)  + q[i+2] + FAST_ROTL64_LO(as_uint2(q[i+3]), 11) + \
                    q[i+4] + FAST_ROTL64_LO(as_uint2(q[i+5]), 27) + q[i+6] + as_ulong(as_uint2(q[i+7]).s10) + \
                    q[i+8] + FAST_ROTL64_HI(as_uint2(q[i+9]), 37) + q[i+10] + FAST_ROTL64_HI(as_uint2(q[i+11]), 43) + \
                    q[i+12] + FAST_ROTL64_HI(as_uint2(q[i+13]), 53) + (SHR(q[i+14],1) ^ q[i+14]) + (SHR(q[i+15],2) ^ q[i+15])

#define SHL(x, n) ((x) << (n))
#define SHR(x, n) ((x) >> (n))

#define s64_0(x)  (SHR((x), 1) ^ SHL((x), 3) ^ FAST_ROTL64_LO(as_uint2((x)),  4) ^ FAST_ROTL64_HI(as_uint2((x)), 37))
#define s64_1(x)  (SHR((x), 1) ^ SHL((x), 2) ^ FAST_ROTL64_LO(as_uint2((x)), 13) ^ FAST_ROTL64_HI(as_uint2((x)), 43))
#define s64_2(x)  (SHR((x), 2) ^ SHL((x), 1) ^ FAST_ROTL64_LO(as_uint2((x)), 19) ^ FAST_ROTL64_HI(as_uint2((x)), 53))
#define s64_3(x)  (SHR((x), 2) ^ SHL((x), 2) ^ FAST_ROTL64_LO(as_uint2((x)), 28) ^ FAST_ROTL64_HI(as_uint2((x)), 59))
#define s64_4(x)  (SHR((x), 1) ^ (x))
#define s64_5(x)  (SHR((x), 2) ^ (x))

#define r64_01(x) FAST_ROTL64_LO(as_uint2((x)),  5)
#define r64_02(x) FAST_ROTL64_LO(as_uint2((x)), 11)
#define r64_03(x) FAST_ROTL64_LO(as_uint2((x)), 27)
#define r64_04(x) devectorize(SWAPDWORDS2(vectorize((x))))
#define r64_05(x) FAST_ROTL64_HI(as_uint2((x)), 37)
#define r64_06(x) FAST_ROTL64_HI(as_uint2((x)), 43)
#define r64_07(x) FAST_ROTL64_HI(as_uint2((x)), 53)

#define Q0	s64_0( (BMW_H[ 5] ^ msg[ 5])-(BMW_H[ 7] ^ msg[ 7])+(BMW_H[10] ^ msg[10])+(BMW_H[13] ^ msg[13])+(BMW_H[14] ^ msg[14])) + BMW_H[1]
#define Q1	s64_1( (BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 8] ^ msg[ 8])+(BMW_H[11] ^ msg[11])+(BMW_H[14] ^ msg[14])-(BMW_H[15] ^ msg[15])) + BMW_H[2]
#define Q2 	s64_2( (BMW_H[ 0] ^ msg[ 0])+(BMW_H[ 7] ^ msg[ 7])+(BMW_H[ 9] ^ msg[ 9])-(BMW_H[12] ^ msg[12])+(BMW_H[15] ^ msg[15])) + BMW_H[3]
#define Q3	s64_3( (BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 1] ^ msg[ 1])+(BMW_H[ 8] ^ msg[ 8])-(BMW_H[10] ^ msg[10])+(BMW_H[13] ^ msg[13])) + BMW_H[4]
#define Q4	s64_4( (BMW_H[ 1] ^ msg[ 1])+(BMW_H[ 2] ^ msg[ 2])+(BMW_H[ 9] ^ msg[ 9])-(BMW_H[11] ^ msg[11])-(BMW_H[14] ^ msg[14])) + BMW_H[5]
#define Q5	s64_0( (BMW_H[ 3] ^ msg[ 3])-(BMW_H[ 2] ^ msg[ 2])+(BMW_H[10] ^ msg[10])-(BMW_H[12] ^ msg[12])+(BMW_H[15] ^ msg[15])) + BMW_H[6]
#define Q6	s64_1( (BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 3] ^ msg[ 3])-(BMW_H[11] ^ msg[11])+(BMW_H[13] ^ msg[13])) + BMW_H[7]
#define Q7	s64_2( (BMW_H[ 1] ^ msg[ 1])-(BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 5] ^ msg[ 5])-(BMW_H[12] ^ msg[12])-(BMW_H[14] ^ msg[14])) + BMW_H[8]
#define Q8	s64_3( (BMW_H[ 2] ^ msg[ 2])-(BMW_H[ 5] ^ msg[ 5])-(BMW_H[ 6] ^ msg[ 6])+(BMW_H[13] ^ msg[13])-(BMW_H[15] ^ msg[15])) + BMW_H[9]
#define Q9	s64_4( (BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 3] ^ msg[ 3])+(BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 7] ^ msg[ 7])+(BMW_H[14] ^ msg[14])) + BMW_H[10]
#define Q10	s64_0( (BMW_H[ 8] ^ msg[ 8])-(BMW_H[ 1] ^ msg[ 1])-(BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 7] ^ msg[ 7])+(BMW_H[15] ^ msg[15])) + BMW_H[11]
#define Q11	s64_1( (BMW_H[ 8] ^ msg[ 8])-(BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 2] ^ msg[ 2])-(BMW_H[ 5] ^ msg[ 5])+(BMW_H[ 9] ^ msg[ 9])) + BMW_H[12]
#define Q12	s64_2( (BMW_H[ 1] ^ msg[ 1])+(BMW_H[ 3] ^ msg[ 3])-(BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 9] ^ msg[ 9])+(BMW_H[10] ^ msg[10])) + BMW_H[13]
#define Q13	s64_3( (BMW_H[ 2] ^ msg[ 2])+(BMW_H[ 4] ^ msg[ 4])+(BMW_H[ 7] ^ msg[ 7])+(BMW_H[10] ^ msg[10])+(BMW_H[11] ^ msg[11])) + BMW_H[14]
#define Q14	s64_4( (BMW_H[ 3] ^ msg[ 3])-(BMW_H[ 5] ^ msg[ 5])+(BMW_H[ 8] ^ msg[ 8])-(BMW_H[11] ^ msg[11])-(BMW_H[12] ^ msg[12])) + BMW_H[15]
#define Q15	s64_0( (BMW_H[12] ^ msg[12])-(BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 9] ^ msg[ 9])+(BMW_H[13] ^ msg[13])) + BMW_H[0]

__device__ __forceinline__ uint64_t BMW_Expand1(uint32_t i, const  uint64_t * msg, const  uint64_t * q, const  uint64_t * h)
{
	return ( s64_1(q[i - 16])          + s64_2(q[i - 15])   + s64_3(q[i - 14]  ) + s64_0(q[i - 13] ) \
           + s64_1(q[i - 12])          + s64_2(q[i - 11])   + s64_3(q[i - 10]  ) + s64_0(q[i -  9] ) \
		   + s64_1(q[i -  8])          + s64_2(q[i -  7])   + s64_3(q[i -  6]  ) + s64_0(q[i -  5] ) \
		   + s64_1(q[i -  4])          + s64_2(q[i -  3])   + s64_3(q[i -  2]  ) + s64_0(q[i -  1] ) \
		   + ((i*(0x0555555555555555ull) + FAST_ROTL64_LO(as_uint2(msg[i - 16]), ((i - 16) + 1)) + FAST_ROTL64_LO(as_uint2(msg[(i-13)]), ((i - 13) + 1)) - FAST_ROTL64_LO(as_uint2(msg[i - 6]), ((i - 6) + 1))) ^ h[((i - 16) + 7)]));
}

__device__ __forceinline__ uint64_t BMW_Expand2(uint32_t i, const uint64_t * msg, const uint64_t * q, const uint64_t * h)
{
	return ( q[i - 16] + r64_01(q[i - 15])  + q[i - 14] + r64_02(q[i - 13]) + \
                    q[i - 12] + r64_03(q[i - 11]) + q[i - 10] + r64_04(q[i - 9]) + \
                    q[i - 8] + r64_05(q[i - 7]) + q[i - 6] + r64_06(q[i - 5]) + \
                    q[i - 4] + r64_07(q[i - 3]) + s64_4(q[i - 2]) + s64_5(q[i - 1]) + \
		   ((i*(0x0555555555555555ull) + FAST_ROTL64_LO(as_uint2(msg[i - 16]), (i - 16) + 1) + FAST_ROTL64_LO(as_uint2(msg[(i - 13) & 15]), ((i - 13) & 15) + 1) - FAST_ROTL64_LO(as_uint2(msg[(i - 6) & 15]), ((i - 6) & 15) + 1)) ^ h[((i - 16) + 7) & 15]));
}

__device__ __forceinline__  void BMW_Compression(uint64_t * msg, const uint64_t *__restrict__ BMW_H, uint64_t *q)
{
	q[ 0] = s64_0( (BMW_H[ 5] ^ msg[ 5])-(BMW_H[ 7] ^ msg[ 7])+(BMW_H[10] ^ msg[10])+(BMW_H[13] ^ msg[13])+(BMW_H[14] ^ msg[14])) + BMW_H[1];
	q[ 1] = s64_1( (BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 8] ^ msg[ 8])+(BMW_H[11] ^ msg[11])+(BMW_H[14] ^ msg[14])-(BMW_H[15] ^ msg[15])) + BMW_H[2];
	q[ 2] = s64_2( (BMW_H[ 0] ^ msg[ 0])+(BMW_H[ 7] ^ msg[ 7])+(BMW_H[ 9] ^ msg[ 9])-(BMW_H[12] ^ msg[12])+(BMW_H[15] ^ msg[15])) + BMW_H[3];
	q[ 3] = s64_3( (BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 1] ^ msg[ 1])+(BMW_H[ 8] ^ msg[ 8])-(BMW_H[10] ^ msg[10])+(BMW_H[13] ^ msg[13])) + BMW_H[4];
	q[ 4] = s64_4( (BMW_H[ 1] ^ msg[ 1])+(BMW_H[ 2] ^ msg[ 2])+(BMW_H[ 9] ^ msg[ 9])-(BMW_H[11] ^ msg[11])-(BMW_H[14] ^ msg[14])) + BMW_H[5];
	q[ 5] = s64_0( (BMW_H[ 3] ^ msg[ 3])-(BMW_H[ 2] ^ msg[ 2])+(BMW_H[10] ^ msg[10])-(BMW_H[12] ^ msg[12])+(BMW_H[15] ^ msg[15])) + BMW_H[6];
	q[ 6] = s64_1( (BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 3] ^ msg[ 3])-(BMW_H[11] ^ msg[11])+(BMW_H[13] ^ msg[13])) + BMW_H[7];
	q[ 7] = s64_2( (BMW_H[ 1] ^ msg[ 1])-(BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 5] ^ msg[ 5])-(BMW_H[12] ^ msg[12])-(BMW_H[14] ^ msg[14])) + BMW_H[8];
	q[ 8] = s64_3( (BMW_H[ 2] ^ msg[ 2])-(BMW_H[ 5] ^ msg[ 5])-(BMW_H[ 6] ^ msg[ 6])+(BMW_H[13] ^ msg[13])-(BMW_H[15] ^ msg[15])) + BMW_H[9];
	q[ 9] = s64_4( (BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 3] ^ msg[ 3])+(BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 7] ^ msg[ 7])+(BMW_H[14] ^ msg[14])) + BMW_H[10];
	q[10] = s64_0( (BMW_H[ 8] ^ msg[ 8])-(BMW_H[ 1] ^ msg[ 1])-(BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 7] ^ msg[ 7])+(BMW_H[15] ^ msg[15])) + BMW_H[11];
	q[11] = s64_1( (BMW_H[ 8] ^ msg[ 8])-(BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 2] ^ msg[ 2])-(BMW_H[ 5] ^ msg[ 5])+(BMW_H[ 9] ^ msg[ 9])) + BMW_H[12];
	q[12] = s64_2( (BMW_H[ 1] ^ msg[ 1])+(BMW_H[ 3] ^ msg[ 3])-(BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 9] ^ msg[ 9])+(BMW_H[10] ^ msg[10])) + BMW_H[13];
	q[13] = s64_3( (BMW_H[ 2] ^ msg[ 2])+(BMW_H[ 4] ^ msg[ 4])+(BMW_H[ 7] ^ msg[ 7])+(BMW_H[10] ^ msg[10])+(BMW_H[11] ^ msg[11])) + BMW_H[14];
	q[14] = s64_4( (BMW_H[ 3] ^ msg[ 3])-(BMW_H[ 5] ^ msg[ 5])+(BMW_H[ 8] ^ msg[ 8])-(BMW_H[11] ^ msg[11])-(BMW_H[12] ^ msg[12])) + BMW_H[15];
	q[15] = s64_0( (BMW_H[12] ^ msg[12])-(BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 9] ^ msg[ 9])+(BMW_H[13] ^ msg[13])) + BMW_H[0];
	
	#pragma unroll 16
	for(int i = 0; i < 16; ++i) q[i + 16] = (i < 2) ? BMW_Expand1(i + 16, msg, q, BMW_H) : BMW_Expand2(i + 16, msg, q, BMW_H);
			
	const ulong XL64 = q[16]^q[17]^q[18]^q[19]^q[20]^q[21]^q[22]^q[23];
	const ulong XH64 = XL64^q[24]^q[25]^q[26]^q[27]^q[28]^q[29]^q[30]^q[31];
		
	msg[0] = (SHL(XH64, 5) ^ SHR(q[16],5) ^ msg[0]) + ( XL64 ^ q[24] ^ q[0]);
	msg[1] = (SHR(XH64, 7) ^ SHL(q[17],8) ^ msg[1]) + ( XL64 ^ q[25] ^ q[1]);
	msg[2] = (SHR(XH64, 5) ^ SHL(q[18],5) ^ msg[2]) + ( XL64 ^ q[26] ^ q[2]);
	msg[3] = (SHR(XH64, 1) ^ SHL(q[19],5) ^ msg[3]) + ( XL64 ^ q[27] ^ q[3]);
	msg[4] = (SHR(XH64, 3) ^ q[20] ^ msg[4]) + ( XL64 ^ q[28] ^ q[4]);
	msg[5] = (SHL(XH64, 6) ^ SHR(q[21],6) ^ msg[5]) + ( XL64 ^ q[29] ^ q[5]);
	msg[6] = (SHR(XH64, 4) ^ SHL(q[22],6) ^ msg[6]) + ( XL64 ^ q[30] ^ q[6]);
	msg[7] = (SHR(XH64,11) ^ SHL(q[23],2) ^ msg[7]) + ( XL64 ^ q[31] ^ q[7]);

	msg[8] = FAST_ROTL64_LO(as_uint2(msg[4]), 9) + ( XH64 ^ q[24] ^ msg[8]) + (SHL(XL64,8) ^ q[23] ^ q[8]);
	msg[9] = FAST_ROTL64_LO(as_uint2(msg[5]),10) + ( XH64 ^ q[25] ^ msg[9]) + (SHR(XL64,6) ^ q[16] ^ q[9]);
	msg[10] = FAST_ROTL64_LO(as_uint2(msg[6]),11) + ( XH64 ^ q[26] ^ msg[10]) + (SHL(XL64,6) ^ q[17] ^ q[10]);
	msg[11] = FAST_ROTL64_LO(as_uint2(msg[7]),12) + ( XH64 ^ q[27] ^ msg[11]) + (SHL(XL64,4) ^ q[18] ^ q[11]);
	msg[12] = FAST_ROTL64_LO(as_uint2(msg[0]),13) + ( XH64 ^ q[28] ^ msg[12]) + (SHR(XL64,3) ^ q[19] ^ q[12]);
	msg[13] = FAST_ROTL64_LO(as_uint2(msg[1]),14) + ( XH64 ^ q[29] ^ msg[13]) + (SHR(XL64,4) ^ q[20] ^ q[13]);
	msg[14] = FAST_ROTL64_LO(as_uint2(msg[2]),15) + ( XH64 ^ q[30] ^ msg[14]) + (SHR(XL64,7) ^ q[21] ^ q[14]);
	msg[15] = FAST_ROTL64_LO(as_uint2(msg[3]),16) + ( XH64 ^ q[31] ^ msg[15]) + (SHR(XL64,2) ^ q[22] ^ q[15]);
}

__constant__ const  uint64_t BMW512_IV[16] =
{
	0x8081828384858687UL, 0x88898A8B8C8D8E8FUL, 0x9091929394959697UL, 0x98999A9B9C9D9E9FUL,
	0xA0A1A2A3A4A5A6A7UL, 0xA8A9AAABACADAEAFUL, 0xB0B1B2B3B4B5B6B7UL, 0xB8B9BABBBCBDBEBFUL,
	0xC0C1C2C3C4C5C6C7UL, 0xC8C9CACBCCCDCECFUL, 0xD0D1D2D3D4D5D6D7UL, 0xD8D9DADBDCDDDEDFUL,
	0xE0E1E2E3E4E5E6E7UL, 0xE8E9EAEBECEDEEEFUL, 0xF0F1F2F3F4F5F6F7UL, 0xF8F9FAFBFCFDFEFFUL
};

__constant__ const  uint64_t BMW512_FINAL[16] =
{
	0xAAAAAAAAAAAAAAA0UL, 0xAAAAAAAAAAAAAAA1UL, 0xAAAAAAAAAAAAAAA2UL, 0xAAAAAAAAAAAAAAA3UL,
	0xAAAAAAAAAAAAAAA4UL, 0xAAAAAAAAAAAAAAA5UL, 0xAAAAAAAAAAAAAAA6UL, 0xAAAAAAAAAAAAAAA7UL,
	0xAAAAAAAAAAAAAAA8UL, 0xAAAAAAAAAAAAAAA9UL, 0xAAAAAAAAAAAAAAAAUL, 0xAAAAAAAAAAAAAAABUL,
	0xAAAAAAAAAAAAAAACUL, 0xAAAAAAAAAAAAAAADUL, 0xAAAAAAAAAAAAAAAEUL, 0xAAAAAAAAAAAAAAAFUL
};

#define GSn4(a,b,c,d,e,f,a1,b1,c1,d1,e1,f1,a2,b2,c2,d2,e2,f2,a3,b3,c3,d3,e3,f3){\
	v[ a] = v[ a] + e + v[ b];		v[a1] = v[a1] + e1 + v[b1];		v[a2] = v[a2] + e2 + v[b2];		v[a3] = v[a3] + e3 + v[b3];\
	v[ d] = SWAPDWORDS2(v[ d] ^ v[ a]);	v[d1] = SWAPDWORDS2(v[d1] ^ v[a1]);	v[d2] = SWAPDWORDS2(v[d2] ^ v[a2]);	v[d3] = SWAPDWORDS2(v[d3] ^ v[a3]);\
	v[ c] = v[ c] + v[ d];			v[c1] = v[c1] + v[d1];			v[c2] = v[c2] + v[d2];			v[c3] = v[c3] + v[d3];\
	v[ b] = ROR2(v[b] ^ v[c],25);		v[b1] = ROR2(v[b1] ^ v[c1],25);		v[b2] = ROR2(v[b2] ^ v[c2],25);		v[b3] = ROR2(v[b3] ^ v[c3],25); \
	v[ a] = v[ a] + f + v[ b];		v[a1] = v[a1] + f1 + v[b1];		v[a2] = v[a2] + f2 + v[b2];		v[a3] = v[a3] + f3 + v[b3];\
	v[ d] = ROR16(v[d] ^ v[a]);		v[d1] = ROR16(v[d1] ^ v[a1]);		v[d2] = ROR16(v[d2] ^ v[a2]);		v[d3] = ROR16(v[d3] ^ v[a3]);\
	v[ c] = v[ c] + v[ d];			v[c1] = v[c1] + v[d1];			v[c2] = v[c2] + v[d2];			v[c3] = v[c3] + v[d3];\
	v[ b] = ROR2(v[b] ^ v[c],11);		v[b1] = ROR2(v[b1] ^ v[c1],11);		v[b2] = ROR2(v[b2] ^ v[c2],11);		v[b3] = ROR2(v[b3] ^ v[c3],11);\
}

__device__ __forceinline__ uint2 cuda_swab64_U2(uint2 a)
{
	// Input:       77665544 33221100
	// Output:      00112233 44556677
	uint2 result;
	result.y = __byte_perm(a.x, 0, 0x0123);
	result.x = __byte_perm(a.y, 0, 0x0123);
	return result;
}

__global__ __launch_bounds__(TPB80,2)
void quark_blake512_gpu_hash_80_bmw_128(const uint32_t threads,const uint32_t startNounce, uint2x4 *const __restrict__ g_hash){
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	uint2 v[16];
	uint2 m[10];
	uint2 xors[16];

	const uint2 h[8] = {
		{0xf3bcc908,0x6a09e667}, {0x84caa73b,0xbb67ae85},{0xfe94f82b,0x3c6ef372}, {0x5f1d36f1,0xa54ff53a},
		{0xade682d1,0x510e527f}, {0x2b3e6c1f,0x9b05688c},{0xfb41bd6b,0x1f83d9ab}, {0x137e2179,0x5be0cd19}
	};
	const uint2 z[16] = {
		{0x85a308d3,0x243f6a88},{0x03707344,0x13198a2e},{0x299f31d0,0xa4093822},{0xec4e6c89,0x082efa98},
		{0x38d01377,0x452821e6},{0x34e90c6c,0xbe5466cf},{0xc97c50dd,0xc0ac29b7},{0xb5470917,0x3f84d5b5},
		{0x8979fb1b,0x9216d5d9},{0x98dfb5ac,0xd1310ba6},{0xd01adfb7,0x2ffd72db},{0x6a267e96,0xb8e1afed},
		{0xf12c7f99,0xba7c9045},{0xb3916cf7,0x24a19947},{0x858efc16,0x0801f2e2},{0x71574e69,0x636920d8}
	};
	const uint32_t m150 = 0x280 ^ z[ 9].x;//make_uint2(0x280,0) ^ z[ 9];//2
	const uint32_t m151 = 0x280 ^ z[13].x;//2
	const uint32_t m152 = 0x280 ^ z[ 8].x;//2
	const uint32_t m153 = 0x280 ^ z[10].x;//2
	const uint32_t m154 = 0x280 ^ z[14].x;//3
	const uint32_t m155 = 0x280 ^ z[ 1].x;//1
	const uint32_t m156 = 0x280 ^ z[ 4].x;//1
	const uint32_t m157 = 0x280 ^ z[ 6].x;//1
	const uint32_t m158 = 0x280 ^ z[11].x;//1

	const uint32_t m130 = 0x01 ^ z[ 6].x;//2
	const uint32_t m131 = 0x01 ^ z[15].x;//2
	const uint32_t m132 = 0x01 ^ z[12].x;//3
	const uint32_t m133 = 0x01 ^ z[ 3].x;//2
	const uint32_t m134 = 0x01 ^ z[ 4].x;//2
	const uint32_t m135 = 0x01 ^ z[14].x;//1
	const uint32_t m136 = 0x01 ^ z[11].x;//1
	const uint32_t m137 = 0x01 ^ z[ 7].x;//1
	const uint32_t m138 = 0x01 ^ z[ 0].x;//1

	const uint32_t m100 = 0x80000000 ^ z[14].y;//4
	const uint32_t m101 = 0x80000000 ^ z[ 5].y;//3
	const uint32_t m102 = 0x80000000 ^ z[15].y;//2
	const uint32_t m103 = 0x80000000 ^ z[ 6].y;//2
	const uint32_t m104 = 0x80000000 ^ z[ 4].y;//1
	const uint32_t m105 = 0x80000000 ^ z[ 2].y;//2
	const uint32_t m106 = 0x80000000 ^ z[11].y;//2

	if (thread < threads){
		int i=0;
		
		#pragma unroll 10
		for (int i=0; i < 10; ++i){
			m[i] = c_m[i];
		}

		m[ 9].x = startNounce + thread;

		#pragma unroll 16
		for(int i=0; i < 16; i++){
			v[i] = c_v[i];
		}

		v[ 0]+= (m[ 9] ^ z[ 8]);
		v[15] = ROR16(v[15] ^ v[ 0]);
		v[10]+= v[15];
		v[ 5] = ROR2(v[ 5] ^ v[10], 11);

		xors[ 0] = z[10];			xors[ 1] = c_x[i++];			xors[ 2] = m[ 9]^z[15];			xors[ 3] = make_uint2(m130,z[ 6].y);
		xors[ 4] = make_uint2(z[14].x,m100);	xors[ 5] = c_x[i++];			xors[ 6] = make_uint2(m150,z[9].y);	xors[ 7] = c_x[i++];
		
		xors[ 8] = c_x[i++];			xors[ 9] = c_x[i++];			xors[10] = z[ 7];			xors[11] = c_x[i++];
		xors[12] = z[ 1];			xors[13] = c_x[i++];			xors[14] = c_x[i++];			xors[15] = c_x[i++];
		
		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);

		//2:{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 }
		xors[ 0] = z[ 8];			xors[ 1] = z[ 0];			xors[ 2] = c_x[i++];			xors[ 3] = make_uint2(m151,z[13].y);
		xors[ 4] = c_x[i++];			xors[ 5] = c_x[i++];			xors[ 6] = c_x[i++];			xors[ 7] = make_uint2(m131,z[15].y);
		
		xors[ 8] = make_uint2(z[14].x,m100);	xors[ 9] = c_x[i++];			xors[10] = c_x[i++];			xors[11] = m[ 9]^z[ 4];
		xors[12] = z[10];			xors[13] = c_x[i++];			xors[14] = c_x[i++];			xors[15] = c_x[i++];
		
		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);

		//3:{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 }
		xors[ 0] = c_x[i++];			xors[ 1] = c_x[i++];			xors[ 2] = make_uint2(m132,z[12].y);	xors[ 3] = z[14];
		xors[ 4] = m[ 9]^z[ 7];			xors[ 5] = c_x[i++];			xors[ 6] = z[13];			xors[ 7] = c_x[i++];
		
		xors[ 8] = c_x[i++];			xors[ 9] = c_x[i++];			xors[10] = c_x[i++];			xors[11] = make_uint2(m152,z[ 8].y);
		xors[12] = c_x[i++];			xors[13] = make_uint2(z[ 5].x,m101);	xors[14] = c_x[i++];			xors[15] = c_x[i++];
		
		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);
		
		//4:{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 }
		xors[ 0] = m[ 9]^z[ 0];			xors[ 1] = c_x[i++];			xors[ 2] = c_x[i++];			xors[ 3] = make_uint2(z[15].x,m102);
		xors[ 4] = c_x[i++];			xors[ 5] = c_x[i++];			xors[ 6] = c_x[i++];			xors[ 7] = make_uint2(m153,z[10].y);
		
		xors[ 8] = z[ 1];			xors[ 9] = z[12];			xors[10] = c_x[i++];			xors[11] = c_x[i++];
		xors[12] = c_x[i++];			xors[13] = z[11];			xors[14] = c_x[i++];			xors[15] = make_uint2(m133,z[ 3].y);
		
		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);
		
		//5:{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
		xors[ 0] = c_x[i++];			xors[ 1] = c_x[i++];			xors[ 2] = c_x[i++];			xors[ 3] = c_x[i++];
		xors[ 4] = z[ 2];			xors[ 5] = make_uint2(z[ 6].x,m103);	xors[ 6] = z[ 0];			xors[ 7] = c_x[i++];
		
		xors[ 8] = c_x[i++];			xors[ 9] = c_x[i++];			xors[10] = make_uint2(m154,z[14].y);	xors[11] = c_x[i++];
		xors[12] = make_uint2(m134,z[ 4].y);	xors[13] = c_x[i++];			xors[14] = z[15];			xors[15] = m[ 9]^z[ 1];
		
		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);
		
		//6:{12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 }
		xors[ 0] = z[ 5];			xors[ 1] = c_x[i++];			xors[ 2] = z[13];			xors[ 3] = c_x[i++];
		xors[ 4] = c_x[i++];			xors[ 5] = make_uint2(m155,z[ 1].y);	xors[ 6] = make_uint2(m135,z[14].y);	xors[ 7] = make_uint2(z[ 4].x,m104);
		
		xors[ 8] = c_x[i++];			xors[ 9] = c_x[i++];			xors[10] = m[ 9]^z[ 2];			xors[11] = c_x[i++];
		xors[12] = c_x[i++];			xors[13] = c_x[i++];			xors[14] = c_x[i++];			xors[15] = z[ 8];
		
		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);

		//7:{13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 }
		xors[ 0] = make_uint2(m136,z[11].y);	xors[ 1] = c_x[i++];			xors[ 2] = z[ 1];			xors[ 3] = c_x[i++];
		xors[ 4] = z[13];			xors[ 5] = z[ 7];			xors[ 6] = c_x[i++];			xors[ 7] = m[ 9]^z[ 3];
		
		xors[ 8] = c_x[i++];			xors[ 9] = make_uint2(m156,z[ 4].y);	xors[10] = c_x[i++];			xors[11] = c_x[i++];
		xors[12] = c_x[i++];			xors[13] = c_x[i++];			xors[14] = c_x[i++];			xors[15] = make_uint2(z[ 2].x,m105);
		
		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);
		
		//8:{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 }
		xors[ 0] = c_x[i++];			xors[ 1] = z[ 9];			xors[ 2] = z[ 3];			xors[ 3] = c_x[i++];
		xors[ 4] = make_uint2(m157,z[ 6].y);	xors[ 5] = m[ 9] ^ z[14];		xors[ 6] = c_x[i++];			xors[ 7] = c_x[i++];

		xors[ 8] = z[ 2];			xors[ 9] = make_uint2(m137,z[ 7].y);	xors[10] = c_x[i++];			xors[11] = make_uint2(z[ 5].x,m101);
		xors[12] = c_x[i++];			xors[13] = c_x[i++];			xors[14] = c_x[i++];			xors[15] = c_x[i++];

		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);

		//9:{10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 }
		xors[ 0] = make_uint2(z[ 2].x,m105);	xors[ 1] = c_x[i++];			xors[ 2] = c_x[i++];			xors[ 3] = c_x[i++];
		xors[ 4] = c_x[i++];			xors[ 5] = c_x[i++];			xors[ 6] = c_x[i++];			xors[ 7] = c_x[i++];
		
		xors[ 8] = make_uint2(m158,z[11].y);	xors[ 9] = m[ 9]^z[14];			xors[10] = c_x[i++];			xors[11] = make_uint2(m138,z[ 0].y);
		xors[12] = z[15];			xors[13] = z[ 9];			xors[14] = z[ 3];			xors[15] = c_x[i++];
		
		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);
		//10:{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 }
		xors[ 0] = c_x[i++];		xors[ 1] = c_x[i++];				xors[ 2] = c_x[i++];			xors[ 3] = c_x[i++];
		xors[ 4] = c_x[i++];		xors[ 5] = c_x[i++];				xors[ 6] = c_x[i++];			xors[ 7] = c_x[i++];
		
		xors[ 8] = c_x[i++];		xors[ 9] = make_uint2(z[11].x,m106);		xors[10] = z[13];			xors[11] = z[15];
		xors[12] = m[ 9]^z[ 8];		xors[13] = z[10];				xors[14] = make_uint2(m132,z[12].y);	xors[15] = make_uint2(m154,z[14].y);
		
		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);
//------------------
		i=0;
		//11:{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 }
		xors[ 0] = z[10];			xors[ 1] = c_x[i++];			xors[ 2] = m[ 9]^z[15];			xors[ 3] = make_uint2(m130,z[ 6].y);
		xors[ 4] = make_uint2(z[14].x,m100);	xors[ 5] = c_x[i++];			xors[ 6] = make_uint2(m150,z[9].y);	xors[ 7] = c_x[i++];
		
		xors[ 8] = c_x[i++];			xors[ 9] = c_x[i++];			xors[10] = z[ 7];			xors[11] = c_x[i++];
		xors[12] = z[ 1];			xors[13] = c_x[i++];			xors[14] = c_x[i++];			xors[15] = c_x[i++];
		
		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);

		//12:{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 }
		xors[ 0] = z[ 8];			xors[ 1] = z[ 0];			xors[ 2] = c_x[i++];			xors[ 3] = make_uint2(m151,z[13].y);
		xors[ 4] = c_x[i++];			xors[ 5] = c_x[i++];			xors[ 6] = c_x[i++];			xors[ 7] = make_uint2(m131,z[15].y);
		
		xors[ 8] = make_uint2(z[14].x,m100);	xors[ 9] = c_x[i++];			xors[10] = c_x[i++];			xors[11] = m[ 9]^z[ 4];
		xors[12] = z[10];			xors[13] = c_x[i++];			xors[14] = c_x[i++];			xors[15] = c_x[i++];
		
		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);

		//13:{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 }
		xors[ 0] = c_x[i++];			xors[ 1] = c_x[i++];			xors[ 2] = make_uint2(m132,z[12].y);	xors[ 3] = z[14];
		xors[ 4] = m[ 9]^z[ 7];			xors[ 5] = c_x[i++];			xors[ 6] = z[13];			xors[ 7] = c_x[i++];
		
		xors[ 8] = c_x[i++];			xors[ 9] = c_x[i++];			xors[10] = c_x[i++];			xors[11] = make_uint2(m152,z[ 8].y);
		xors[12] = c_x[i++];			xors[13] = make_uint2(z[ 5].x,m101);	xors[14] = c_x[i++];			xors[15] = c_x[i++];
		
		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);

		//14:{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 }
		xors[ 0] = m[ 9]^z[ 0];			xors[ 1] = c_x[i++];			xors[ 2] = c_x[i++];			xors[ 3] = make_uint2(z[15].x,m102);
		xors[ 4] = c_x[i++];			xors[ 5] = c_x[i++];			xors[ 6] = c_x[i++];			xors[ 7] = make_uint2(m153,z[10].y);
		
		xors[ 8] = z[ 1];			xors[ 9] = z[12];			xors[10] = c_x[i++];			xors[11] = c_x[i++];
		xors[12] = c_x[i++];			xors[13] = z[11];			xors[14] = c_x[i++];			xors[15] = make_uint2(m133,z[ 3].y);
		
		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);
		//15:{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
		xors[ 0] = c_x[i++];			xors[ 1] = c_x[i++];			xors[ 2] = c_x[i++];			xors[ 3] = c_x[i++];
		xors[ 4] = z[ 2];			xors[ 5] = make_uint2(z[ 6].x,m103);	xors[ 6] = z[ 0];			xors[ 7] = c_x[i++];
		
		xors[ 8] = c_x[i++];			xors[ 9] = c_x[i++];			xors[10] = make_uint2(m154,z[14].y);	xors[11] = c_x[i++];
		xors[12] = make_uint2(m134,z[ 4].y);	xors[13] = c_x[i++];			xors[14] = z[15];			xors[15] = m[ 9]^z[ 1];
		
		GSn4(0, 4, 8,12, xors[ 0],xors[ 4], 1, 5, 9,13, xors[ 1],xors[ 5], 2, 6,10,14, xors[ 2],xors[ 6], 3, 7,11,15, xors[ 3],xors[ 7]);
		GSn4(0, 5,10,15, xors[ 8],xors[12], 1, 6,11,12, xors[ 9],xors[13], 2, 7, 8,13, xors[10],xors[14], 3, 4, 9,14, xors[11],xors[15]);

		v[0] = cuda_swab64_U2(xor3x(v[0],h[0],v[ 8]));
		v[1] = cuda_swab64_U2(xor3x(v[1],h[1],v[ 9]));
		v[2] = cuda_swab64_U2(xor3x(v[2],h[2],v[10]));
		v[3] = cuda_swab64_U2(xor3x(v[3],h[3],v[11]));
		v[4] = cuda_swab64_U2(xor3x(v[4],h[4],v[12]));
		v[5] = cuda_swab64_U2(xor3x(v[5],h[5],v[13]));
		v[6] = cuda_swab64_U2(xor3x(v[6],h[6],v[14]));
		v[7] = cuda_swab64_U2(xor3x(v[7],h[7],v[15]));

		uint64_t msg[8];
		uint64_t msg0[16] = { 0 }, msg1[16] = { 0 };
		uint64_t q[32];
		
#pragma unroll 8
		for(int i=0;i<8;i++)msg[i]=devectorize(v[i]);

#pragma unroll 8
		for(int i = 0; i < 8; ++i) msg0[i] = (msg[i]);

		msg1[0] = 0x80UL;
		msg1[15] = 1024UL;
		BMW_Compression(msg0, BMW512_IV, q);
		BMW_Compression(msg1, msg0,q);
		BMW_Compression(msg1, BMW512_FINAL,q);

		uint2x4* outpt = &g_hash[thread<<1];
		outpt[0] = *(uint2x4*)&msg1[8];
		outpt[1] = *(uint2x4*)&msg1[12];
	}
}

__host__
void quark_blake512_cpu_hash_80_bmw_128(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash){
	dim3 grid((threads + TPB80-1)/TPB80);
	dim3 block(TPB80);

	quark_blake512_gpu_hash_80_bmw_128<<<grid, block>>>(threads, startNounce, (uint2x4*)d_outputHash);
}