#include "hip/hip_runtime.h"
// Auf QuarkCoin spezialisierte Version von Groestl inkl. Bitslice

#include <stdio.h>
#include <memory.h>
#include <sys/types.h> // off_t

#include <cuda_helper.h>
#include <cuda_vectors.h>
#include <cuda_vectors_alexis.h>

#ifdef __INTELLISENSE__
#define __CUDA_ARCH__ 500
#endif

#define TPB 256
#define THF 4U

#if __CUDA_ARCH__ >= 300
#include "groestl_functions_quad.h"
#include "groestl_transf_quad.h"
#endif

#define WANT_GROESTL80
#ifdef WANT_GROESTL80
__constant__ static uint32_t c_Message80[20];
#endif

#include "cuda_quark_groestl512_sm2.cuh"

__global__ __launch_bounds__(TPB, THF)
void quark_groestl512_gpu_hash_64_quad(const uint32_t threads, const uint32_t startNounce, uint32_t * g_hash, uint32_t * __restrict g_nonceVector)
{
#if __CUDA_ARCH__ >= 300

	// BEWARE : 4-WAY CODE (one hash need 4 threads)
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;

	if (thread < threads)
	{
		uint32_t message[8];
		uint32_t state[8];

		uint32_t nounce = g_nonceVector ? g_nonceVector[thread] : (startNounce + thread);
		off_t hashPosition = nounce - startNounce;
		uint32_t *pHash = &g_hash[hashPosition << 4];

		const uint32_t thr = threadIdx.x & 0x3; // % THF

		/*| M0 M1 M2 M3 | M4 M5 M6 M7 | (input)
		--|-------------|-------------|
		T0|  0  4  8 12 | 80          |
		T1|  1  5    13 |             |
		T2|  2  6    14 |             |
		T3|  3  7    15 |          01 |
		--|-------------|-------------| */

		#pragma unroll
		for(int k=0;k<4;k++) message[k] = pHash[thr + (k * THF)];

		#pragma unroll
		for(int k=4;k<8;k++) message[k] = 0;

		if (thr == 0) message[4] = 0x80U; // end of data tag
		if (thr == 3) message[7] = 0x01000000U;

		uint32_t msgBitsliced[8];
		to_bitslice_quad(message, msgBitsliced);

		groestl512_progressMessage_quad(state, msgBitsliced);

		uint32_t hash[16];
		from_bitslice_quad(state, hash);

		// uint4 = 4x4 uint32_t = 16 bytes
		if (thr == 0) {
			uint4 *phash = (uint4*) hash;
			uint4 *outpt = (uint4*) pHash;
			outpt[0] = phash[0];
			outpt[1] = phash[1];
			outpt[2] = phash[2];
			outpt[3] = phash[3];
		}
	}
#endif
}

__host__
void quark_groestl512_cpu_init(int thr_id, uint32_t threads)
{
	int dev_id = device_map[thr_id];
	cuda_get_arch(thr_id);
	if (device_sm[dev_id] < 300 || cuda_arch[dev_id] < 300)
		quark_groestl512_sm20_init(thr_id, threads);
}

__host__
void quark_groestl512_cpu_free(int thr_id)
{
	int dev_id = device_map[thr_id];
	if (device_sm[dev_id] < 300 || cuda_arch[dev_id] < 300)
		quark_groestl512_sm20_free(thr_id);
}

__host__
void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	uint32_t threadsperblock = TPB;

	// Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
	// mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
	const uint32_t factor = THF;

	dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
	dim3 block(threadsperblock);

	int dev_id = device_map[thr_id];

	if (device_sm[dev_id] >= 300 && cuda_arch[dev_id] >= 300)
		quark_groestl512_gpu_hash_64_quad<<<grid, block>>>(threads, startNounce, d_hash, d_nonceVector);
	else
		quark_groestl512_sm20_hash_64(thr_id, threads, startNounce, d_nonceVector, d_hash, order);
}

// --------------------------------------------------------------------------------------------------------------------------------------------

#ifdef WANT_GROESTL80

__host__
void groestl512_setBlock_80(int thr_id, uint32_t *endiandata)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_Message80), endiandata, sizeof(c_Message80), 0, hipMemcpyHostToDevice);
}

__global__ __launch_bounds__(TPB, THF)
void groestl512_gpu_hash_80_quad(const uint32_t threads, const uint32_t startNounce, uint32_t * g_outhash)
{
#if __CUDA_ARCH__ >= 300
	// BEWARE : 4-WAY CODE (one hash need 4 threads)
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;
	if (thread < threads)
	{
		const uint32_t thr = threadIdx.x & 0x3; // % THF

		/*| M0 M1 M2 M3 M4 | M5 M6 M7 | (input)
		--|----------------|----------|
		T0|  0  4  8 12 16 | 80       |
		T1|  1  5       17 |          |
		T2|  2  6       18 |          |
		T3|  3  7       Nc |       01 |
		--|----------------|----------| TPR */

		uint32_t message[8];

		#pragma unroll 5
		for(int k=0; k<5; k++) message[k] = c_Message80[thr + (k * THF)];

		#pragma unroll 3
		for(int k=5; k<8; k++) message[k] = 0;

		if (thr == 0) message[5] = 0x80U;
		if (thr == 3) {
			message[4] = cuda_swab32(startNounce + thread);
			message[7] = 0x01000000U;
		}

		uint32_t msgBitsliced[8];
		to_bitslice_quad(message, msgBitsliced);

		uint32_t state[8];
		groestl512_progressMessage_quad(state, msgBitsliced);

		uint32_t hash[16];
		from_bitslice_quad(state, hash);

		if (thr == 0) { /* 4 threads were done */
			const off_t hashPosition = thread;
			//if (!thread) hash[15] = 0xFFFFFFFF;
			uint4 *outpt = (uint4*) &g_outhash[hashPosition << 4];
			uint4 *phash = (uint4*) hash;
			outpt[0] = phash[0];
			outpt[1] = phash[1];
			outpt[2] = phash[2];
			outpt[3] = phash[3];
		}
	}
#endif
}

__host__
void groestl512_cuda_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash)
{
	int dev_id = device_map[thr_id];

	if (device_sm[dev_id] >= 300 && cuda_arch[dev_id] >= 300) {
		const uint32_t threadsperblock = TPB;
		const uint32_t factor = THF;

		dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
		dim3 block(threadsperblock);

		groestl512_gpu_hash_80_quad <<<grid, block>>> (threads, startNounce, d_hash);

	} else {

		const uint32_t threadsperblock = 256;
		dim3 grid((threads + threadsperblock-1)/threadsperblock);
		dim3 block(threadsperblock);

		groestl512_gpu_hash_80_sm2 <<<grid, block>>> (threads, startNounce, d_hash);
	}
}

#endif

#define TPB128 512

__global__ __launch_bounds__(TPB128, 2)
void quark_groestl512_gpu_hash_128_quad(const uint32_t threads,  uint32_t * g_hash)
{
#if __CUDA_ARCH__ >= 300

	// BEWARE : 4-WAY CODE (one hash need 4 threads)
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;

	if (thread < threads)
	{
		uint32_t m[8];
		uint32_t g[8];
		uint32_t h[8];
		uint32_t m2[8];

		uint32_t *pHash = &g_hash[thread << 4];

		const uint32_t thr = threadIdx.x & 0x3; // % THF

		/*| M0 M1 M2 M3 | M4 M5 M6 M7 | (input)
		--|-------------|-------------|
		T0|  0  4  8 12 | 80          |
		T1|  1  5    13 |             |
		T2|  2  6    14 |             |
		T3|  3  7    15 |          01 |
		--|-------------|-------------| */

		#pragma unroll
		for(int k=0;k<4;k++) m[k] = pHash[thr + (k * THF)];

		#pragma unroll
		for(int k=4;k<8;k++) m[k] = 0;

		if (thr == 0) m[4] = 0x0U; // end of data tag
		if (thr == 3) m[7] = 0x0000000U;

		uint32_t mBitsliced[8];
		to_bitslice_quad(m, mBitsliced);

		#pragma unroll
		for(int k=0;k<4;k++) g[k] = pHash[thr + (k * THF)];

		#pragma unroll
		for(int k=4;k<8;k++) g[k] = 0;

		if (thr == 0) g[4] = 0x0U; // end of data tag
		if (thr == 3) g[7] = 0x0020000U;

		uint32_t gBitsliced[8];
		to_bitslice_quad(g, gBitsliced);

		#pragma unroll
		for(int k=0;k<4;k++) h[k] = 0;

		#pragma unroll
		for(int k=4;k<8;k++) h[k] = 0;

		if (thr == 0) h[4] = 0x0U; // end of data tag
		if (thr == 3) h[7] = 0x0020000U;

		uint32_t hBitsliced[8];
		to_bitslice_quad(h, hBitsliced);

		#pragma unroll
		for(int k=0;k<8;k++) m2[k] = 0;

		if (thr == 0) m2[0] = 0x80U; // end of data tag
		if (thr == 3) m2[7] = 0x2000000;

		uint32_t m2Bitsliced[8];
		to_bitslice_quad(m2, m2Bitsliced);

        groestl512_perm_P_quad(gBitsliced);
		groestl512_perm_Q_quad(mBitsliced);

		for (unsigned int u = 0; u < 8; u++){
			hBitsliced[u] ^= gBitsliced[u] ^ mBitsliced[u];
		}

		for (unsigned int u = 0; u < 8; u++){
			gBitsliced[u] = m2Bitsliced[u] ^ hBitsliced[u];
		}
        groestl512_perm_P_quad(gBitsliced);

		if(thr == 0){
			m2Bitsliced[0]=0x89aecd65;
			m2Bitsliced[1]=0x64a6d130;
			m2Bitsliced[2]=0x3f3d9e18;
			m2Bitsliced[3]=0xae0389d4;
			m2Bitsliced[4]=0xbbf2c8a2;
			m2Bitsliced[5]=0x3b1b2f4;
			m2Bitsliced[6]=0xeca737be;
			m2Bitsliced[7]=0xe4d92093;
		}

		if(thr == 1){
			m2Bitsliced[0]=0x813d1bbf;
			m2Bitsliced[1]=0x64aea6;
			m2Bitsliced[2]=0xcac17604;
			m2Bitsliced[3]=0x7edc9d98;
			m2Bitsliced[4]=0xf895469;
			m2Bitsliced[5]=0x3450f60c;
			m2Bitsliced[6]=0xedaae1a4;
			m2Bitsliced[7]=0x363761e9;
		}

		if(thr == 2){
			m2Bitsliced[0]=0xb81a7b17;
			m2Bitsliced[1]=0x322e9ee6;
			m2Bitsliced[2]=0x1ce5c5cd;
			m2Bitsliced[3]=0x79e2d9b0;
			m2Bitsliced[4]=0x7734ec9c;
			m2Bitsliced[5]=0xde433ef;
			m2Bitsliced[6]=0x7459f800;
			m2Bitsliced[7]=0xec98575b;
		}

		if(thr == 3){
			m2Bitsliced[0]=0xe1eb324e;
			m2Bitsliced[1]=0x30530c30;
			m2Bitsliced[2]=0xef2f21d0;
			m2Bitsliced[3]=0x8a0194b3;
			m2Bitsliced[4]=0x9516fd30;
			m2Bitsliced[5]=0xd8f3a4bf;
			m2Bitsliced[6]=0x4d3cbccd;
			m2Bitsliced[7]=0x5ac552f;
		}

        uint32_t hxBitsliced[8];

		for (unsigned int u = 0; u < 8; u++){
			hxBitsliced[u] = hBitsliced[u] ^= gBitsliced[u] ^ m2Bitsliced[u];
		}

        groestl512_perm_P_quad(hxBitsliced);

		for (unsigned int u = 0; u < 8; u++){
			hxBitsliced[u] ^= hBitsliced[u];
		}

		uint32_t hash[16];
		from_bitslice_quad(hxBitsliced, hash);

		if (thr == 0) {
			uint2x4 *phash = (uint2x4*) hash;
			uint2x4 *outpt = (uint2x4*) pHash;
			outpt[0] = phash[0];
			outpt[1] = phash[1];
		}
	}
#endif
}

__host__
void quark_groestl512_cpu_hash_128(int thr_id, uint32_t threads,  uint32_t *d_hash)
{
	uint32_t threadsperblock = TPB128;
	const uint32_t factor = THF;

	dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
	dim3 block(threadsperblock);
	quark_groestl512_gpu_hash_128_quad<<<grid, block>>>(threads, d_hash);
}