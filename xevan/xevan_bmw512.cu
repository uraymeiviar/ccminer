#include "hip/hip_runtime.h"

#include "cuda_helper_alexis.h"
#include "cuda_vectors_alexis.h"

#define sph_u64 uint64_t

#define TPB_BMW 256
#define BMW_SH 64
// Wolf's BMW512, loosely based on SPH's implementation
#define as_uint2(x) (x)
#define FAST_ROTL64_LO ROTL64
#define FAST_ROTL64_HI ROTL64

#undef CONST_EXP2

#define CONST_EXP2  q[i+0] + FAST_ROTL64_LO(as_uint2(q[i+1]), 5)  + q[i+2] + FAST_ROTL64_LO(as_uint2(q[i+3]), 11) + \
                    q[i+4] + FAST_ROTL64_LO(as_uint2(q[i+5]), 27) + q[i+6] + as_ulong(as_uint2(q[i+7]).s10) + \
                    q[i+8] + FAST_ROTL64_HI(as_uint2(q[i+9]), 37) + q[i+10] + FAST_ROTL64_HI(as_uint2(q[i+11]), 43) + \
                    q[i+12] + FAST_ROTL64_HI(as_uint2(q[i+13]), 53) + (SHR(q[i+14],1) ^ q[i+14]) + (SHR(q[i+15],2) ^ q[i+15])

#undef SHL
#undef SHR
#define SHL(x, n) ((x) << (n))
#define SHR(x, n) ((x) >> (n))

#define s64_0(x)  (SHR((x), 1) ^ SHL((x), 3) ^ FAST_ROTL64_LO(as_uint2((x)),  4) ^ FAST_ROTL64_HI(as_uint2((x)), 37))
#define s64_1(x)  (SHR((x), 1) ^ SHL((x), 2) ^ FAST_ROTL64_LO(as_uint2((x)), 13) ^ FAST_ROTL64_HI(as_uint2((x)), 43))
#define s64_2(x)  (SHR((x), 2) ^ SHL((x), 1) ^ FAST_ROTL64_LO(as_uint2((x)), 19) ^ FAST_ROTL64_HI(as_uint2((x)), 53))
#define s64_3(x)  (SHR((x), 2) ^ SHL((x), 2) ^ FAST_ROTL64_LO(as_uint2((x)), 28) ^ FAST_ROTL64_HI(as_uint2((x)), 59))
#define s64_4(x)  (SHR((x), 1) ^ (x))
#define s64_5(x)  (SHR((x), 2) ^ (x))

#define r64_01(x) FAST_ROTL64_LO(as_uint2((x)),  5)
#define r64_02(x) FAST_ROTL64_LO(as_uint2((x)), 11)
#define r64_03(x) FAST_ROTL64_LO(as_uint2((x)), 27)
#define r64_04(x) devectorize(SWAPDWORDS2(vectorize((x))))
#define r64_05(x) FAST_ROTL64_HI(as_uint2((x)), 37)
#define r64_06(x) FAST_ROTL64_HI(as_uint2((x)), 43)
#define r64_07(x) FAST_ROTL64_HI(as_uint2((x)), 53)

#define Q0	s64_0( (BMW_H[ 5] ^ msg[ 5])-(BMW_H[ 7] ^ msg[ 7])+(BMW_H[10] ^ msg[10])+(BMW_H[13] ^ msg[13])+(BMW_H[14] ^ msg[14])) + BMW_H[1]
#define Q1	s64_1( (BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 8] ^ msg[ 8])+(BMW_H[11] ^ msg[11])+(BMW_H[14] ^ msg[14])-(BMW_H[15] ^ msg[15])) + BMW_H[2]
#define Q2 	s64_2( (BMW_H[ 0] ^ msg[ 0])+(BMW_H[ 7] ^ msg[ 7])+(BMW_H[ 9] ^ msg[ 9])-(BMW_H[12] ^ msg[12])+(BMW_H[15] ^ msg[15])) + BMW_H[3]
#define Q3	s64_3( (BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 1] ^ msg[ 1])+(BMW_H[ 8] ^ msg[ 8])-(BMW_H[10] ^ msg[10])+(BMW_H[13] ^ msg[13])) + BMW_H[4]
#define Q4	s64_4( (BMW_H[ 1] ^ msg[ 1])+(BMW_H[ 2] ^ msg[ 2])+(BMW_H[ 9] ^ msg[ 9])-(BMW_H[11] ^ msg[11])-(BMW_H[14] ^ msg[14])) + BMW_H[5]
#define Q5	s64_0( (BMW_H[ 3] ^ msg[ 3])-(BMW_H[ 2] ^ msg[ 2])+(BMW_H[10] ^ msg[10])-(BMW_H[12] ^ msg[12])+(BMW_H[15] ^ msg[15])) + BMW_H[6]
#define Q6	s64_1( (BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 3] ^ msg[ 3])-(BMW_H[11] ^ msg[11])+(BMW_H[13] ^ msg[13])) + BMW_H[7]
#define Q7	s64_2( (BMW_H[ 1] ^ msg[ 1])-(BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 5] ^ msg[ 5])-(BMW_H[12] ^ msg[12])-(BMW_H[14] ^ msg[14])) + BMW_H[8]
#define Q8	s64_3( (BMW_H[ 2] ^ msg[ 2])-(BMW_H[ 5] ^ msg[ 5])-(BMW_H[ 6] ^ msg[ 6])+(BMW_H[13] ^ msg[13])-(BMW_H[15] ^ msg[15])) + BMW_H[9]
#define Q9	s64_4( (BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 3] ^ msg[ 3])+(BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 7] ^ msg[ 7])+(BMW_H[14] ^ msg[14])) + BMW_H[10]
#define Q10	s64_0( (BMW_H[ 8] ^ msg[ 8])-(BMW_H[ 1] ^ msg[ 1])-(BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 7] ^ msg[ 7])+(BMW_H[15] ^ msg[15])) + BMW_H[11]
#define Q11	s64_1( (BMW_H[ 8] ^ msg[ 8])-(BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 2] ^ msg[ 2])-(BMW_H[ 5] ^ msg[ 5])+(BMW_H[ 9] ^ msg[ 9])) + BMW_H[12]
#define Q12	s64_2( (BMW_H[ 1] ^ msg[ 1])+(BMW_H[ 3] ^ msg[ 3])-(BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 9] ^ msg[ 9])+(BMW_H[10] ^ msg[10])) + BMW_H[13]
#define Q13	s64_3( (BMW_H[ 2] ^ msg[ 2])+(BMW_H[ 4] ^ msg[ 4])+(BMW_H[ 7] ^ msg[ 7])+(BMW_H[10] ^ msg[10])+(BMW_H[11] ^ msg[11])) + BMW_H[14]
#define Q14	s64_4( (BMW_H[ 3] ^ msg[ 3])-(BMW_H[ 5] ^ msg[ 5])+(BMW_H[ 8] ^ msg[ 8])-(BMW_H[11] ^ msg[11])-(BMW_H[12] ^ msg[12])) + BMW_H[15]
#define Q15	s64_0( (BMW_H[12] ^ msg[12])-(BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 9] ^ msg[ 9])+(BMW_H[13] ^ msg[13])) + BMW_H[0]

__device__ __forceinline__ uint64_t BMW_Expand1(uint32_t i, const  uint64_t * msg, const  uint64_t * q, const  uint64_t * h)
{
	return ( s64_1(q[i - 16])          + s64_2(q[i - 15])   + s64_3(q[i - 14]  ) + s64_0(q[i - 13] ) \
           + s64_1(q[i - 12])          + s64_2(q[i - 11])   + s64_3(q[i - 10]  ) + s64_0(q[i -  9] ) \
		   + s64_1(q[i -  8])          + s64_2(q[i -  7])   + s64_3(q[i -  6]  ) + s64_0(q[i -  5] ) \
		   + s64_1(q[i -  4])          + s64_2(q[i -  3])   + s64_3(q[i -  2]  ) + s64_0(q[i -  1] ) \
		   + ((i*(0x0555555555555555ull) + FAST_ROTL64_LO(as_uint2(msg[i - 16]), ((i - 16) + 1)) + FAST_ROTL64_LO(as_uint2(msg[(i-13)]), ((i - 13) + 1)) - FAST_ROTL64_LO(as_uint2(msg[i - 6]), ((i - 6) + 1))) ^ h[((i - 16) + 7)]));
}

__device__ __forceinline__ uint64_t BMW_Expand2(uint32_t i, const uint64_t * msg, const uint64_t * q, const uint64_t * h)
{
	return ( q[i - 16] + r64_01(q[i - 15])  + q[i - 14] + r64_02(q[i - 13]) + \
                    q[i - 12] + r64_03(q[i - 11]) + q[i - 10] + r64_04(q[i - 9]) + \
                    q[i - 8] + r64_05(q[i - 7]) + q[i - 6] + r64_06(q[i - 5]) + \
                    q[i - 4] + r64_07(q[i - 3]) + s64_4(q[i - 2]) + s64_5(q[i - 1]) + \
		   ((i*(0x0555555555555555ull) + FAST_ROTL64_LO(as_uint2(msg[i - 16]), (i - 16) + 1) + FAST_ROTL64_LO(as_uint2(msg[(i - 13) & 15]), ((i - 13) & 15) + 1) - FAST_ROTL64_LO(as_uint2(msg[(i - 6) & 15]), ((i - 6) & 15) + 1)) ^ h[((i - 16) + 7) & 15]));
}

__device__ __forceinline__  void BMW_Compression(uint64_t * msg, const uint64_t *__restrict__ BMW_H, uint64_t *q)
{
	q[ 0] = s64_0( (BMW_H[ 5] ^ msg[ 5])-(BMW_H[ 7] ^ msg[ 7])+(BMW_H[10] ^ msg[10])+(BMW_H[13] ^ msg[13])+(BMW_H[14] ^ msg[14])) + BMW_H[1];
	q[ 1] = s64_1( (BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 8] ^ msg[ 8])+(BMW_H[11] ^ msg[11])+(BMW_H[14] ^ msg[14])-(BMW_H[15] ^ msg[15])) + BMW_H[2];
	q[ 2] = s64_2( (BMW_H[ 0] ^ msg[ 0])+(BMW_H[ 7] ^ msg[ 7])+(BMW_H[ 9] ^ msg[ 9])-(BMW_H[12] ^ msg[12])+(BMW_H[15] ^ msg[15])) + BMW_H[3];
	q[ 3] = s64_3( (BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 1] ^ msg[ 1])+(BMW_H[ 8] ^ msg[ 8])-(BMW_H[10] ^ msg[10])+(BMW_H[13] ^ msg[13])) + BMW_H[4];
	q[ 4] = s64_4( (BMW_H[ 1] ^ msg[ 1])+(BMW_H[ 2] ^ msg[ 2])+(BMW_H[ 9] ^ msg[ 9])-(BMW_H[11] ^ msg[11])-(BMW_H[14] ^ msg[14])) + BMW_H[5];
	q[ 5] = s64_0( (BMW_H[ 3] ^ msg[ 3])-(BMW_H[ 2] ^ msg[ 2])+(BMW_H[10] ^ msg[10])-(BMW_H[12] ^ msg[12])+(BMW_H[15] ^ msg[15])) + BMW_H[6];
	q[ 6] = s64_1( (BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 3] ^ msg[ 3])-(BMW_H[11] ^ msg[11])+(BMW_H[13] ^ msg[13])) + BMW_H[7];
	q[ 7] = s64_2( (BMW_H[ 1] ^ msg[ 1])-(BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 5] ^ msg[ 5])-(BMW_H[12] ^ msg[12])-(BMW_H[14] ^ msg[14])) + BMW_H[8];
	q[ 8] = s64_3( (BMW_H[ 2] ^ msg[ 2])-(BMW_H[ 5] ^ msg[ 5])-(BMW_H[ 6] ^ msg[ 6])+(BMW_H[13] ^ msg[13])-(BMW_H[15] ^ msg[15])) + BMW_H[9];
	q[ 9] = s64_4( (BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 3] ^ msg[ 3])+(BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 7] ^ msg[ 7])+(BMW_H[14] ^ msg[14])) + BMW_H[10];
	q[10] = s64_0( (BMW_H[ 8] ^ msg[ 8])-(BMW_H[ 1] ^ msg[ 1])-(BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 7] ^ msg[ 7])+(BMW_H[15] ^ msg[15])) + BMW_H[11];
	q[11] = s64_1( (BMW_H[ 8] ^ msg[ 8])-(BMW_H[ 0] ^ msg[ 0])-(BMW_H[ 2] ^ msg[ 2])-(BMW_H[ 5] ^ msg[ 5])+(BMW_H[ 9] ^ msg[ 9])) + BMW_H[12];
	q[12] = s64_2( (BMW_H[ 1] ^ msg[ 1])+(BMW_H[ 3] ^ msg[ 3])-(BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 9] ^ msg[ 9])+(BMW_H[10] ^ msg[10])) + BMW_H[13];
	q[13] = s64_3( (BMW_H[ 2] ^ msg[ 2])+(BMW_H[ 4] ^ msg[ 4])+(BMW_H[ 7] ^ msg[ 7])+(BMW_H[10] ^ msg[10])+(BMW_H[11] ^ msg[11])) + BMW_H[14];
	q[14] = s64_4( (BMW_H[ 3] ^ msg[ 3])-(BMW_H[ 5] ^ msg[ 5])+(BMW_H[ 8] ^ msg[ 8])-(BMW_H[11] ^ msg[11])-(BMW_H[12] ^ msg[12])) + BMW_H[15];
	q[15] = s64_0( (BMW_H[12] ^ msg[12])-(BMW_H[ 4] ^ msg[ 4])-(BMW_H[ 6] ^ msg[ 6])-(BMW_H[ 9] ^ msg[ 9])+(BMW_H[13] ^ msg[13])) + BMW_H[0];
	
	#pragma unroll 16
	for(int i = 0; i < 16; ++i) q[i + 16] = (i < 2) ? BMW_Expand1(i + 16, msg, q, BMW_H) : BMW_Expand2(i + 16, msg, q, BMW_H);
			
	const ulong XL64 = q[16]^q[17]^q[18]^q[19]^q[20]^q[21]^q[22]^q[23];
	const ulong XH64 = XL64^q[24]^q[25]^q[26]^q[27]^q[28]^q[29]^q[30]^q[31];
		
	msg[0] = (SHL(XH64, 5) ^ SHR(q[16],5) ^ msg[0]) + ( XL64 ^ q[24] ^ q[0]);
	msg[1] = (SHR(XH64, 7) ^ SHL(q[17],8) ^ msg[1]) + ( XL64 ^ q[25] ^ q[1]);
	msg[2] = (SHR(XH64, 5) ^ SHL(q[18],5) ^ msg[2]) + ( XL64 ^ q[26] ^ q[2]);
	msg[3] = (SHR(XH64, 1) ^ SHL(q[19],5) ^ msg[3]) + ( XL64 ^ q[27] ^ q[3]);
	msg[4] = (SHR(XH64, 3) ^ q[20] ^ msg[4]) + ( XL64 ^ q[28] ^ q[4]);
	msg[5] = (SHL(XH64, 6) ^ SHR(q[21],6) ^ msg[5]) + ( XL64 ^ q[29] ^ q[5]);
	msg[6] = (SHR(XH64, 4) ^ SHL(q[22],6) ^ msg[6]) + ( XL64 ^ q[30] ^ q[6]);
	msg[7] = (SHR(XH64,11) ^ SHL(q[23],2) ^ msg[7]) + ( XL64 ^ q[31] ^ q[7]);

	msg[8] = FAST_ROTL64_LO(as_uint2(msg[4]), 9) + ( XH64 ^ q[24] ^ msg[8]) + (SHL(XL64,8) ^ q[23] ^ q[8]);
	msg[9] = FAST_ROTL64_LO(as_uint2(msg[5]),10) + ( XH64 ^ q[25] ^ msg[9]) + (SHR(XL64,6) ^ q[16] ^ q[9]);
	msg[10] = FAST_ROTL64_LO(as_uint2(msg[6]),11) + ( XH64 ^ q[26] ^ msg[10]) + (SHL(XL64,6) ^ q[17] ^ q[10]);
	msg[11] = FAST_ROTL64_LO(as_uint2(msg[7]),12) + ( XH64 ^ q[27] ^ msg[11]) + (SHL(XL64,4) ^ q[18] ^ q[11]);
	msg[12] = FAST_ROTL64_LO(as_uint2(msg[0]),13) + ( XH64 ^ q[28] ^ msg[12]) + (SHR(XL64,3) ^ q[19] ^ q[12]);
	msg[13] = FAST_ROTL64_LO(as_uint2(msg[1]),14) + ( XH64 ^ q[29] ^ msg[13]) + (SHR(XL64,4) ^ q[20] ^ q[13]);
	msg[14] = FAST_ROTL64_LO(as_uint2(msg[2]),15) + ( XH64 ^ q[30] ^ msg[14]) + (SHR(XL64,7) ^ q[21] ^ q[14]);
	msg[15] = FAST_ROTL64_LO(as_uint2(msg[3]),16) + ( XH64 ^ q[31] ^ msg[15]) + (SHR(XL64,2) ^ q[22] ^ q[15]);
}

__global__ __launch_bounds__(TPB_BMW,2)
void xevan_bmw512_gpu_hash_64x(uint32_t threads, uint64_t *const __restrict__ g_hash, const uint32_t *const __restrict__ g_nonceVector){
	const  uint64_t BMW512_IV[16] =
	{
		0x8081828384858687UL, 0x88898A8B8C8D8E8FUL, 0x9091929394959697UL, 0x98999A9B9C9D9E9FUL,
		0xA0A1A2A3A4A5A6A7UL, 0xA8A9AAABACADAEAFUL, 0xB0B1B2B3B4B5B6B7UL, 0xB8B9BABBBCBDBEBFUL,
		0xC0C1C2C3C4C5C6C7UL, 0xC8C9CACBCCCDCECFUL, 0xD0D1D2D3D4D5D6D7UL, 0xD8D9DADBDCDDDEDFUL,
		0xE0E1E2E3E4E5E6E7UL, 0xE8E9EAEBECEDEEEFUL, 0xF0F1F2F3F4F5F6F7UL, 0xF8F9FAFBFCFDFEFFUL
	};

	const  uint64_t BMW512_FINAL[16] =
	{
		0xAAAAAAAAAAAAAAA0UL, 0xAAAAAAAAAAAAAAA1UL, 0xAAAAAAAAAAAAAAA2UL, 0xAAAAAAAAAAAAAAA3UL,
		0xAAAAAAAAAAAAAAA4UL, 0xAAAAAAAAAAAAAAA5UL, 0xAAAAAAAAAAAAAAA6UL, 0xAAAAAAAAAAAAAAA7UL,
		0xAAAAAAAAAAAAAAA8UL, 0xAAAAAAAAAAAAAAA9UL, 0xAAAAAAAAAAAAAAAAUL, 0xAAAAAAAAAAAAAAABUL,
		0xAAAAAAAAAAAAAAACUL, 0xAAAAAAAAAAAAAAADUL, 0xAAAAAAAAAAAAAAAEUL, 0xAAAAAAAAAAAAAAAFUL
	};

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads){
		const uint32_t hashPosition = (g_nonceVector == NULL) ? thread : g_nonceVector[thread];
		uint64_t *inpHash = &g_hash[8 * hashPosition];
		uint64_t msg[8];

		uint2x4* phash = (uint2x4*)inpHash;
		uint2x4* outpt = (uint2x4*)msg;
		outpt[0] = __ldg4(&phash[0]);
		outpt[1] = __ldg4(&phash[1]);

		uint64_t msg0[16] = { 0 }, msg1[16] = { 0 };
		uint64_t q[32];

		for(int i = 0; i < 8; ++i) msg0[i] = (msg[i]);

		msg1[0] = 0x80UL;
		msg1[15] = 1024UL;

		BMW_Compression(msg0, BMW512_IV, q);
		BMW_Compression(msg1, msg0,q);
		BMW_Compression(msg1, BMW512_FINAL,q);
	
		for(int i = 0; i < 8; ++i) inpHash[i] = (msg1[i + 8]);
	}
}

__host__ void xevan_bmw512_cpu_hash_64x(int thr_id, uint32_t threads, uint32_t *d_nonceVector, uint32_t *d_hash)
{
    const uint32_t threadsperblock = TPB_BMW;

    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    xevan_bmw512_gpu_hash_64x<<<grid, block>>>(threads, (uint64_t*)d_hash, d_nonceVector);
}