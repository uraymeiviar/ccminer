#include "hip/hip_runtime.h"
#include "cuda_helper.h"
#include "cuda_helper_alexis.h"
#include "cuda_vectors_alexis.h"

#define F1(x6, x5, x4, x3, x2, x1, x0) \
	(((x1) & ((x0) ^ (x4))) ^ ((x2) & (x5)) ^ ((x3) & (x6)) ^ (x0))

#define F2(x6, x5, x4, x3, x2, x1, x0) \
	(((x2) & (((x1) & ~(x3)) ^ ((x4) & (x5)) ^ (x6) ^ (x0))) ^ ((x4) & ((x1) ^ (x5))) ^ ((x3 & (x5)) ^ (x0)))

#define F3(x6, x5, x4, x3, x2, x1, x0) \
	(((x3) & (((x1) & (x2)) ^ (x6) ^ (x0))) ^ ((x1) & (x4)) ^ ((x2) & (x5)) ^ (x0))

#define F4(x6, x5, x4, x3, x2, x1, x0) \
	(((x3) & (((x1) & (x2)) ^ ((x4) | (x6)) ^ (x5))) ^ ((x4) & ((~(x2) & (x5)) ^ (x1) ^ (x6) ^ (x0))) ^ ((x2) & (x6)) ^ (x0))

#define F5(x6, x5, x4, x3, x2, x1, x0) \
	(((x0) & ~(((x1) & (x2) & (x3)) ^ (x5))) ^ ((x1) & (x4)) ^ ((x2) & (x5)) ^ ((x3) & (x6)))

#define FP5_1(x6, x5, x4, x3, x2, x1, x0) \
	F1(x3, x4, x1, x0, x5, x2, x6)
#define FP5_2(x6, x5, x4, x3, x2, x1, x0) \
	F2(x6, x2, x1, x0, x3, x4, x5)
#define FP5_3(x6, x5, x4, x3, x2, x1, x0) \
	F3(x2, x6, x0, x4, x3, x1, x5)
#define FP5_4(x6, x5, x4, x3, x2, x1, x0) \
	F4(x1, x5, x3, x2, x0, x4, x6)
#define FP5_5(x6, x5, x4, x3, x2, x1, x0) \
    F5(x2, x5, x0, x6, x4, x3, x1)
    
#define STEP(n, p, x7, x6, x5, x4, x3, x2, x1, x0, w, c) { \
	uint32_t t = FP ## n ## _ ## p(x6, x5, x4, x3, x2, x1, x0); \
	(x7) = (uint32_t)(ROTR32(t, 7) + ROTR32((x7), 11) + (w) + (c)); \
}

#define PASS1(n, in) { \
	STEP(n, 1, s7, s6, s5, s4, s3, s2, s1, s0, in[ 0], 0U); \
	STEP(n, 1, s6, s5, s4, s3, s2, s1, s0, s7, in[ 1], 0U); \
	STEP(n, 1, s5, s4, s3, s2, s1, s0, s7, s6, in[ 2], 0U); \
	STEP(n, 1, s4, s3, s2, s1, s0, s7, s6, s5, in[ 3], 0U); \
	STEP(n, 1, s3, s2, s1, s0, s7, s6, s5, s4, in[ 4], 0U); \
	STEP(n, 1, s2, s1, s0, s7, s6, s5, s4, s3, in[ 5], 0U); \
	STEP(n, 1, s1, s0, s7, s6, s5, s4, s3, s2, in[ 6], 0U); \
	STEP(n, 1, s0, s7, s6, s5, s4, s3, s2, s1, in[ 7], 0U); \
 \
	STEP(n, 1, s7, s6, s5, s4, s3, s2, s1, s0, in[ 8], 0U); \
	STEP(n, 1, s6, s5, s4, s3, s2, s1, s0, s7, in[ 9], 0U); \
	STEP(n, 1, s5, s4, s3, s2, s1, s0, s7, s6, in[10], 0U); \
	STEP(n, 1, s4, s3, s2, s1, s0, s7, s6, s5, in[11], 0U); \
	STEP(n, 1, s3, s2, s1, s0, s7, s6, s5, s4, in[12], 0U); \
	STEP(n, 1, s2, s1, s0, s7, s6, s5, s4, s3, in[13], 0U); \
	STEP(n, 1, s1, s0, s7, s6, s5, s4, s3, s2, in[14], 0U); \
	STEP(n, 1, s0, s7, s6, s5, s4, s3, s2, s1, in[15], 0U); \
 \
	STEP(n, 1, s7, s6, s5, s4, s3, s2, s1, s0, in[16], 0U); \
	STEP(n, 1, s6, s5, s4, s3, s2, s1, s0, s7, in[17], 0U); \
	STEP(n, 1, s5, s4, s3, s2, s1, s0, s7, s6, in[18], 0U); \
	STEP(n, 1, s4, s3, s2, s1, s0, s7, s6, s5, in[19], 0U); \
	STEP(n, 1, s3, s2, s1, s0, s7, s6, s5, s4, in[20], 0U); \
	STEP(n, 1, s2, s1, s0, s7, s6, s5, s4, s3, in[21], 0U); \
	STEP(n, 1, s1, s0, s7, s6, s5, s4, s3, s2, in[22], 0U); \
	STEP(n, 1, s0, s7, s6, s5, s4, s3, s2, s1, in[23], 0U); \
 \
	STEP(n, 1, s7, s6, s5, s4, s3, s2, s1, s0, in[24], 0U); \
	STEP(n, 1, s6, s5, s4, s3, s2, s1, s0, s7, in[25], 0U); \
	STEP(n, 1, s5, s4, s3, s2, s1, s0, s7, s6, in[26], 0U); \
	STEP(n, 1, s4, s3, s2, s1, s0, s7, s6, s5, in[27], 0U); \
	STEP(n, 1, s3, s2, s1, s0, s7, s6, s5, s4, in[28], 0U); \
	STEP(n, 1, s2, s1, s0, s7, s6, s5, s4, s3, in[29], 0U); \
	STEP(n, 1, s1, s0, s7, s6, s5, s4, s3, s2, in[30], 0U); \
	STEP(n, 1, s0, s7, s6, s5, s4, s3, s2, s1, in[31], 0U); \
}

#define PASS2(n, in) { \
	STEP(n, 2, s7, s6, s5, s4, s3, s2, s1, s0, in[ 5], 0x452821E6); \
	STEP(n, 2, s6, s5, s4, s3, s2, s1, s0, s7, in[14], 0x38D01377); \
	STEP(n, 2, s5, s4, s3, s2, s1, s0, s7, s6, in[26], 0xBE5466CF); \
	STEP(n, 2, s4, s3, s2, s1, s0, s7, s6, s5, in[18], 0x34E90C6C); \
	STEP(n, 2, s3, s2, s1, s0, s7, s6, s5, s4, in[11], 0xC0AC29B7); \
	STEP(n, 2, s2, s1, s0, s7, s6, s5, s4, s3, in[28], 0xC97C50DD); \
	STEP(n, 2, s1, s0, s7, s6, s5, s4, s3, s2, in[ 7], 0x3F84D5B5); \
	STEP(n, 2, s0, s7, s6, s5, s4, s3, s2, s1, in[16], 0xB5470917); \
 \
	STEP(n, 2, s7, s6, s5, s4, s3, s2, s1, s0, in[ 0], 0x9216D5D9); \
	STEP(n, 2, s6, s5, s4, s3, s2, s1, s0, s7, in[23], 0x8979FB1B); \
	STEP(n, 2, s5, s4, s3, s2, s1, s0, s7, s6, in[20], 0xD1310BA6); \
	STEP(n, 2, s4, s3, s2, s1, s0, s7, s6, s5, in[22], 0x98DFB5AC); \
	STEP(n, 2, s3, s2, s1, s0, s7, s6, s5, s4, in[ 1], 0x2FFD72DB); \
	STEP(n, 2, s2, s1, s0, s7, s6, s5, s4, s3, in[10], 0xD01ADFB7); \
	STEP(n, 2, s1, s0, s7, s6, s5, s4, s3, s2, in[ 4], 0xB8E1AFED); \
	STEP(n, 2, s0, s7, s6, s5, s4, s3, s2, s1, in[ 8], 0x6A267E96); \
 \
	STEP(n, 2, s7, s6, s5, s4, s3, s2, s1, s0, in[30], 0xBA7C9045); \
	STEP(n, 2, s6, s5, s4, s3, s2, s1, s0, s7, in[ 3], 0xF12C7F99); \
	STEP(n, 2, s5, s4, s3, s2, s1, s0, s7, s6, in[21], 0x24A19947); \
	STEP(n, 2, s4, s3, s2, s1, s0, s7, s6, s5, in[ 9], 0xB3916CF7); \
	STEP(n, 2, s3, s2, s1, s0, s7, s6, s5, s4, in[17], 0x0801F2E2); \
	STEP(n, 2, s2, s1, s0, s7, s6, s5, s4, s3, in[24], 0x858EFC16); \
	STEP(n, 2, s1, s0, s7, s6, s5, s4, s3, s2, in[29], 0x636920D8); \
	STEP(n, 2, s0, s7, s6, s5, s4, s3, s2, s1, in[ 6], 0x71574E69); \
 \
	STEP(n, 2, s7, s6, s5, s4, s3, s2, s1, s0, in[19], 0xA458FEA3); \
	STEP(n, 2, s6, s5, s4, s3, s2, s1, s0, s7, in[12], 0xF4933D7E); \
	STEP(n, 2, s5, s4, s3, s2, s1, s0, s7, s6, in[15], 0x0D95748F); \
	STEP(n, 2, s4, s3, s2, s1, s0, s7, s6, s5, in[13], 0x728EB658); \
	STEP(n, 2, s3, s2, s1, s0, s7, s6, s5, s4, in[ 2], 0x718BCD58); \
	STEP(n, 2, s2, s1, s0, s7, s6, s5, s4, s3, in[25], 0x82154AEE); \
	STEP(n, 2, s1, s0, s7, s6, s5, s4, s3, s2, in[31], 0x7B54A41D); \
	STEP(n, 2, s0, s7, s6, s5, s4, s3, s2, s1, in[27], 0xC25A59B5); \
}

#define PASS3(n, in) { \
	STEP(n, 3, s7, s6, s5, s4, s3, s2, s1, s0, in[19], 0x9C30D539); \
	STEP(n, 3, s6, s5, s4, s3, s2, s1, s0, s7, in[ 9], 0x2AF26013); \
	STEP(n, 3, s5, s4, s3, s2, s1, s0, s7, s6, in[ 4], 0xC5D1B023); \
	STEP(n, 3, s4, s3, s2, s1, s0, s7, s6, s5, in[20], 0x286085F0); \
	STEP(n, 3, s3, s2, s1, s0, s7, s6, s5, s4, in[28], 0xCA417918); \
	STEP(n, 3, s2, s1, s0, s7, s6, s5, s4, s3, in[17], 0xB8DB38EF); \
	STEP(n, 3, s1, s0, s7, s6, s5, s4, s3, s2, in[ 8], 0x8E79DCB0); \
	STEP(n, 3, s0, s7, s6, s5, s4, s3, s2, s1, in[22], 0x603A180E); \
 \
	STEP(n, 3, s7, s6, s5, s4, s3, s2, s1, s0, in[29], 0x6C9E0E8B); \
	STEP(n, 3, s6, s5, s4, s3, s2, s1, s0, s7, in[14], 0xB01E8A3E); \
	STEP(n, 3, s5, s4, s3, s2, s1, s0, s7, s6, in[25], 0xD71577C1); \
	STEP(n, 3, s4, s3, s2, s1, s0, s7, s6, s5, in[12], 0xBD314B27); \
	STEP(n, 3, s3, s2, s1, s0, s7, s6, s5, s4, in[24], 0x78AF2FDA); \
	STEP(n, 3, s2, s1, s0, s7, s6, s5, s4, s3, in[30], 0x55605C60); \
	STEP(n, 3, s1, s0, s7, s6, s5, s4, s3, s2, in[16], 0xE65525F3); \
	STEP(n, 3, s0, s7, s6, s5, s4, s3, s2, s1, in[26], 0xAA55AB94); \
 \
	STEP(n, 3, s7, s6, s5, s4, s3, s2, s1, s0, in[31], 0x57489862); \
	STEP(n, 3, s6, s5, s4, s3, s2, s1, s0, s7, in[15], 0x63E81440); \
	STEP(n, 3, s5, s4, s3, s2, s1, s0, s7, s6, in[ 7], 0x55CA396A); \
	STEP(n, 3, s4, s3, s2, s1, s0, s7, s6, s5, in[ 3], 0x2AAB10B6); \
	STEP(n, 3, s3, s2, s1, s0, s7, s6, s5, s4, in[ 1], 0xB4CC5C34); \
	STEP(n, 3, s2, s1, s0, s7, s6, s5, s4, s3, in[ 0], 0x1141E8CE); \
	STEP(n, 3, s1, s0, s7, s6, s5, s4, s3, s2, in[18], 0xA15486AF); \
	STEP(n, 3, s0, s7, s6, s5, s4, s3, s2, s1, in[27], 0x7C72E993); \
 \
	STEP(n, 3, s7, s6, s5, s4, s3, s2, s1, s0, in[13], 0xB3EE1411); \
	STEP(n, 3, s6, s5, s4, s3, s2, s1, s0, s7, in[ 6], 0x636FBC2A); \
	STEP(n, 3, s5, s4, s3, s2, s1, s0, s7, s6, in[21], 0x2BA9C55D); \
	STEP(n, 3, s4, s3, s2, s1, s0, s7, s6, s5, in[10], 0x741831F6); \
	STEP(n, 3, s3, s2, s1, s0, s7, s6, s5, s4, in[23], 0xCE5C3E16); \
	STEP(n, 3, s2, s1, s0, s7, s6, s5, s4, s3, in[11], 0x9B87931E); \
	STEP(n, 3, s1, s0, s7, s6, s5, s4, s3, s2, in[ 5], 0xAFD6BA33); \
	STEP(n, 3, s0, s7, s6, s5, s4, s3, s2, s1, in[ 2], 0x6C24CF5C); \
}

#define PASS4(n, in) { \
	STEP(n, 4, s7, s6, s5, s4, s3, s2, s1, s0, in[24], 0x7A325381); \
	STEP(n, 4, s6, s5, s4, s3, s2, s1, s0, s7, in[ 4], 0x28958677); \
	STEP(n, 4, s5, s4, s3, s2, s1, s0, s7, s6, in[ 0], 0x3B8F4898); \
	STEP(n, 4, s4, s3, s2, s1, s0, s7, s6, s5, in[14], 0x6B4BB9AF); \
	STEP(n, 4, s3, s2, s1, s0, s7, s6, s5, s4, in[ 2], 0xC4BFE81B); \
	STEP(n, 4, s2, s1, s0, s7, s6, s5, s4, s3, in[ 7], 0x66282193); \
	STEP(n, 4, s1, s0, s7, s6, s5, s4, s3, s2, in[28], 0x61D809CC); \
	STEP(n, 4, s0, s7, s6, s5, s4, s3, s2, s1, in[23], 0xFB21A991); \
 \
	STEP(n, 4, s7, s6, s5, s4, s3, s2, s1, s0, in[26], 0x487CAC60); \
	STEP(n, 4, s6, s5, s4, s3, s2, s1, s0, s7, in[ 6], 0x5DEC8032); \
	STEP(n, 4, s5, s4, s3, s2, s1, s0, s7, s6, in[30], 0xEF845D5D); \
	STEP(n, 4, s4, s3, s2, s1, s0, s7, s6, s5, in[20], 0xE98575B1); \
	STEP(n, 4, s3, s2, s1, s0, s7, s6, s5, s4, in[18], 0xDC262302); \
	STEP(n, 4, s2, s1, s0, s7, s6, s5, s4, s3, in[25], 0xEB651B88); \
	STEP(n, 4, s1, s0, s7, s6, s5, s4, s3, s2, in[19], 0x23893E81); \
	STEP(n, 4, s0, s7, s6, s5, s4, s3, s2, s1, in[ 3], 0xD396ACC5); \
 \
	STEP(n, 4, s7, s6, s5, s4, s3, s2, s1, s0, in[22], 0x0F6D6FF3); \
	STEP(n, 4, s6, s5, s4, s3, s2, s1, s0, s7, in[11], 0x83F44239); \
	STEP(n, 4, s5, s4, s3, s2, s1, s0, s7, s6, in[31], 0x2E0B4482); \
	STEP(n, 4, s4, s3, s2, s1, s0, s7, s6, s5, in[21], 0xA4842004); \
	STEP(n, 4, s3, s2, s1, s0, s7, s6, s5, s4, in[ 8], 0x69C8F04A); \
	STEP(n, 4, s2, s1, s0, s7, s6, s5, s4, s3, in[27], 0x9E1F9B5E); \
	STEP(n, 4, s1, s0, s7, s6, s5, s4, s3, s2, in[12], 0x21C66842); \
	STEP(n, 4, s0, s7, s6, s5, s4, s3, s2, s1, in[ 9], 0xF6E96C9A); \
 \
	STEP(n, 4, s7, s6, s5, s4, s3, s2, s1, s0, in[ 1], 0x670C9C61); \
	STEP(n, 4, s6, s5, s4, s3, s2, s1, s0, s7, in[29], 0xABD388F0); \
	STEP(n, 4, s5, s4, s3, s2, s1, s0, s7, s6, in[ 5], 0x6A51A0D2); \
	STEP(n, 4, s4, s3, s2, s1, s0, s7, s6, s5, in[15], 0xD8542F68); \
	STEP(n, 4, s3, s2, s1, s0, s7, s6, s5, s4, in[17], 0x960FA728); \
	STEP(n, 4, s2, s1, s0, s7, s6, s5, s4, s3, in[10], 0xAB5133A3); \
	STEP(n, 4, s1, s0, s7, s6, s5, s4, s3, s2, in[16], 0x6EEF0B6C); \
	STEP(n, 4, s0, s7, s6, s5, s4, s3, s2, s1, in[13], 0x137A3BE4); \
}

#define PASS5(n, in) { \
	STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[27], 0xBA3BF050); \
	STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[ 3], 0x7EFB2A98); \
	STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[21], 0xA1F1651D); \
	STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[26], 0x39AF0176); \
	STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[17], 0x66CA593E); \
	STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[11], 0x82430E88); \
	STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[20], 0x8CEE8619); \
	STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[29], 0x456F9FB4); \
 \
	STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[19], 0x7D84A5C3); \
	STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[ 0], 0x3B8B5EBE); \
	STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[12], 0xE06F75D8); \
	STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[ 7], 0x85C12073); \
	STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[13], 0x401A449F); \
	STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[ 8], 0x56C16AA6); \
	STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[31], 0x4ED3AA62); \
	STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[10], 0x363F7706); \
 \
	STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[ 5], 0x1BFEDF72); \
	STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[ 9], 0x429B023D); \
	STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[14], 0x37D0D724); \
	STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[30], 0xD00A1248); \
	STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[18], 0xDB0FEAD3); \
	STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[ 6], 0x49F1C09B); \
	STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[28], 0x075372C9); \
	STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[24], 0x80991B7B); \
 \
	STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[ 2], 0x25D479D8); \
	STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[23], 0xF6E8DEF7); \
	STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[16], 0xE3FE501A); \
	STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[22], 0xB6794C3B); \
	STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[ 4], 0x976CE0BD); \
	STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[ 1], 0x04C006BA); \
	STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[25], 0xC1A94FB6); \
	STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[15], 0x409F60C4); \
}

#define PASS5_final(n, in) { \
	STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[27], 0xBA3BF050); \
	STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[ 3], 0x7EFB2A98); \
	STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[21], 0xA1F1651D); \
	STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[26], 0x39AF0176); \
	STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[17], 0x66CA593E); \
	STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[11], 0x82430E88); \
	STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[20], 0x8CEE8619); \
	STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[29], 0x456F9FB4); \
 \
	STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[19], 0x7D84A5C3); \
	STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[ 0], 0x3B8B5EBE); \
	STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[12], 0xE06F75D8); \
	STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[ 7], 0x85C12073); \
	STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[13], 0x401A449F); \
	STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[ 8], 0x56C16AA6); \
	STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[31], 0x4ED3AA62); \
	STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[10], 0x363F7706); \
 \
	STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[ 5], 0x1BFEDF72); \
	STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[ 9], 0x429B023D); \
	STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[14], 0x37D0D724); \
	STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[30], 0xD00A1248); \
	STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[18], 0xDB0FEAD3); \
	STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[ 6], 0x49F1C09B); \
	STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[28], 0x075372C9); \
	STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[24], 0x80991B7B); \
 \
	STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[ 2], 0x25D479D8); \
	STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[23], 0xF6E8DEF7); \
}

#define sph_u32 uint32_t

#define H_SAVE_STATE \
	sph_u32 u0, u1, u2, u3, u4, u5, u6, u7; \
	do { \
		u0 = s0; \
		u1 = s1; \
		u2 = s2; \
		u3 = s3; \
		u4 = s4; \
		u5 = s5; \
		u6 = s6; \
		u7 = s7; \
	} while (0)

#define H_UPDATE_STATE   do { \
		s0 = SPH_T32(s0 + u0); \
		s1 = SPH_T32(s1 + u1); \
		s2 = SPH_T32(s2 + u2); \
		s3 = SPH_T32(s3 + u3); \
		s4 = SPH_T32(s4 + u4); \
		s5 = SPH_T32(s5 + u5); \
		s6 = SPH_T32(s6 + u6); \
		s7 = SPH_T32(s7 + u7); \
	} while (0)

#define CORE5(in)  do { \
	H_SAVE_STATE; \
	PASS1(5, in); \
	PASS2(5, in); \
	PASS3(5, in); \
	PASS4(5, in); \
	PASS5(5, in); \
	H_UPDATE_STATE; \
} while (0)

#define CORE5_F(in)  do { \
	H_SAVE_STATE; \
	PASS1(5, in); \
	PASS2(5, in); \
	PASS3(5, in); \
	PASS4(5, in); \
	PASS5_final(5, in); \
	H_UPDATE_STATE; \
} while (0)

#define TPB 512

__global__ __launch_bounds__(TPB, 2)
void xevan_haval512_gpu_hash_128(const uint32_t threads,const uint64_t*  g_hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads)
	{
        uint32_t *pHash = (uint32_t *)&g_hash[thread<<3];
// haval
		sph_u32 s0 = SPH_C32(0x243F6A88);
		sph_u32 s1 = SPH_C32(0x85A308D3);
		sph_u32 s2 = SPH_C32(0x13198A2E);
		sph_u32 s3 = SPH_C32(0x03707344);
		sph_u32 s4 = SPH_C32(0xA4093822);
		sph_u32 s5 = SPH_C32(0x299F31D0);
		sph_u32 s6 = SPH_C32(0x082EFA98);
		sph_u32 s7 = SPH_C32(0xEC4E6C89);

		sph_u32 X_var[32];

		uint2x4* phash = (uint2x4*)pHash;
		uint2x4* outpt = (uint2x4*)X_var;
		outpt[0] = __ldg4(&phash[0]);
		outpt[1] = __ldg4(&phash[1]);

		#pragma unroll 16
		for (int i = 16; i < 32; i++){
			X_var[i] = 0;
		}

  		CORE5(X_var);

  		X_var[0] = 0x00000001U;

		#pragma unroll 28
		for (int i = 1; i < 29; i++){
			X_var[i] = 0;
		}

		X_var[29] = 0x40290000U;
		X_var[30] = 0x00000400U;
		X_var[31] = 0x00000000U;

		CORE5(X_var);

		pHash[0] = s0;
		pHash[1] = s1;
		pHash[2] = s2;
		pHash[3] = s3;
		pHash[4] = s4;
		pHash[5] = s5;
		pHash[6] = s6;
		pHash[7] = s7;

		pHash[8] = 0;
		pHash[9] = 0;
		pHash[10] = 0;
		pHash[11] = 0;
		pHash[12] = 0;
		pHash[13] = 0;
		pHash[14] = 0;
		pHash[15] = 0;
	}
}

__host__
void xevan_haval512_cpu_hash_128(int thr_id, uint32_t threads, uint32_t *d_hash)
{
	dim3 grid((threads + TPB-1)/TPB);
	dim3 block(TPB);
	xevan_haval512_gpu_hash_128 <<<grid, block>>> (threads, (uint64_t*)d_hash);
}

#define TPB_F 512

__global__ __launch_bounds__(TPB_F, 4)
void xevan_haval512_gpu_hash_128_final(const uint32_t threads,const uint64_t* __restrict__ g_hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads)
	{
		uint32_t *pHash = (uint32_t *)&g_hash[thread<<3];
		// haval
		sph_u32 s0 = SPH_C32(0x243F6A88);
		sph_u32 s1 = SPH_C32(0x85A308D3);
		sph_u32 s2 = SPH_C32(0x13198A2E);
		sph_u32 s3 = SPH_C32(0x03707344);
		sph_u32 s4 = SPH_C32(0xA4093822);
		sph_u32 s5 = SPH_C32(0x299F31D0);
		sph_u32 s6 = SPH_C32(0x082EFA98);
		sph_u32 s7 = SPH_C32(0xEC4E6C89);

  		sph_u32 X_var[32];

		uint2x4* phash = (uint2x4*)pHash;
		uint2x4* outpt = (uint2x4*)X_var;
		outpt[0] = __ldg4(&phash[0]);
		outpt[1] = __ldg4(&phash[1]);

		//#pragma unroll 16
		for (int i = 16; i < 32; i++){
			X_var[i] = 0;
		}

		CORE5(X_var);

		X_var[0] = 0x00000001U;

		#pragma unroll 28
		for (int i = 1; i < 29; i++){
			X_var[i] = 0;
		}
		X_var[29] = 0x40290000U;
		X_var[30] = 0x00000400U;
		X_var[31] = 0x00000000U;

		CORE5_F(X_var);

		pHash[0] = s0;
		pHash[1] = s1;
		pHash[2] = s2;
		pHash[3] = s3;
		pHash[4] = s4;
		pHash[5] = s5;
		pHash[6] = s6;
		pHash[7] = s7;

		pHash[8] = 0;
		pHash[9] = 0;
		pHash[10] = 0;
		pHash[11] = 0;
		pHash[12] = 0;
		pHash[13] = 0;
		pHash[14] = 0;
		pHash[15] = 0;
	}
}

__host__
void xevan_haval512_cpu_hash_128_final(int thr_id, uint32_t threads, uint32_t *d_hash)
{
	dim3 grid((threads + TPB_F-1)/TPB_F);
	dim3 block(TPB_F);

	xevan_haval512_gpu_hash_128_final <<<grid, block>>> (threads, (uint64_t*)d_hash);
}