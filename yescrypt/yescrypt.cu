#include "hip/hip_runtime.h"
#include "miner.h"
#include "cuda_helper.h"
extern "C" {
#include "sph/yescrypt.h"
}

#include <math.h>
extern "C" {
#include "SHA3api_ref.h"
}

extern void yescrypt_cpu_init(int thr_id, int threads, uint32_t *d_hash1, uint32_t *d_hash2, uint32_t *d_hash3, uint32_t *d_hash4);
extern void yescrypt_setTarget(int thr_id, uint32_t pdata[20], const char *key, uint32_t key_len);
extern void yescrypt_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *resultnonces, uint32_t target, const uint32_t N, const uint32_t r, const uint32_t p);
extern void yescrypt_cpu_free(int thr_id);

extern char *yescrypt_key;
extern size_t yescrypt_key_len;
extern uint32_t yescrypt_param_N;
extern uint32_t yescrypt_param_r;
extern uint32_t yescrypt_param_p;

const char *client_key;    // true for GlobalBoost-Y
size_t client_key_len = 0xff;
static bool init[MAX_GPUS] = { 0 };

void yescrypt_hash_base(void *state, const void *input, const uint32_t N, const uint32_t r, const uint32_t p, const char *key, const size_t key_len)
{
	if (client_key_len == 0xff)
	{
		client_key = key;
		client_key_len = key_len;
	}
	yescrypt_bsty((unsigned char*)input, 80, (unsigned char*)input, 80, N, r, p, (unsigned char *)state, 32);
}

void yescrypt_hash(void *state, const void *input)
{
	yescrypt_hash_base(state, input, 2048, 8, 1, NULL, 0);
}

void yescryptr8_hash(void *state, const void *input)
{
	yescrypt_hash_base(state, input, 2048, 8, 1, "Client Key", 10);
}

void yescryptr16_hash(void *state, const void *input)
{
	yescrypt_hash_base(state, input, 4096, 16, 1, "Client Key", 10);
}

void yescryptr16v2_hash(void *state, const void *input)
{
	yescrypt_hash_base(state, input, 4096, 16, 4, "PPTPPubKey", 10);
}

void yescryptr32_hash(void *state, const void *input)
{
	yescrypt_hash_base(state, input, 4096, 32, 1, "WaviBanana", 10);
}

int scanhash_yescrypt_base(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done,
	const uint32_t N, const uint32_t r, const uint32_t p,
	const char *key, const size_t key_len) {
	static __thread uint32_t *d_hash1 = NULL;
	static __thread uint32_t *d_hash2 = NULL;
	static __thread uint32_t *d_hash3 = NULL;
	static __thread uint32_t *d_hash4 = NULL;
	uint32_t *pdata = work->data;
    uint32_t *ptarget = work->target;

	const uint32_t first_nonce = pdata[19];

	int dev_id = device_map[thr_id];
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, dev_id);

	uint32_t CUDAcore_count;
	if (device_sm[dev_id] == 600)		// Pascal(P100)
		CUDAcore_count = props.multiProcessorCount * 64;
	else if (device_sm[dev_id] >= 500)	// Maxwell/Pascal(other)/Volta
		CUDAcore_count = props.multiProcessorCount * 128;
	else if (device_sm[dev_id] >= 300)	// Kepler
		CUDAcore_count = props.multiProcessorCount * 96; // * 192
	else if (device_sm[dev_id] >= 210)	// Fermi(GF11x)
		CUDAcore_count = props.multiProcessorCount * 48;
	else					// Fermi(GF10x)
		CUDAcore_count = props.multiProcessorCount * 32;

	uint32_t throughputmax;
#if defined WIN32 && !defined _WIN64
	// 2GB limit for hipMalloc
	uint32_t max_thread_multiple = (min(0x7fffffffULL, props.totalGlobalMem) - 256 * 1024 * 1024) / (((520 + 2 * r * (N + 16 * p)) * sizeof(uint32_t)) * CUDAcore_count);
#else
	uint32_t max_thread_multiple = (props.totalGlobalMem - 256 * 1024 * 1024) / (((520 + 2 * r * (N + 16 * p)) * sizeof(uint32_t)) * CUDAcore_count);
#endif

	if (device_sm[dev_id] > 500)		// Maxwell(GTX9xx)/Pascal/Volta
		throughputmax = device_intensity(dev_id, __func__, CUDAcore_count * min(3, max_thread_multiple));
	else if (device_sm[dev_id] == 500)	// Maxwell(GTX750Ti/GTX750)
		throughputmax = device_intensity(dev_id, __func__, CUDAcore_count * min(2, max_thread_multiple));
	else if (device_sm[dev_id] >= 300)	// Kepler
		throughputmax = device_intensity(dev_id, __func__, CUDAcore_count);
	else if (device_sm[dev_id] >= 210)	// Fermi(GF11x)
		throughputmax = device_intensity(dev_id, __func__, CUDAcore_count);
	else								// Fermi(GF10x)
		throughputmax = device_intensity(dev_id, __func__, CUDAcore_count);

	throughputmax = (throughputmax / CUDAcore_count) * CUDAcore_count;
	if (throughputmax == 0) throughputmax = CUDAcore_count;

	uint32_t throughput = min(throughputmax, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00ff;

	if (!init[thr_id])
	{
		applog(LOG_WARNING, "Using intensity %.3f (%d threads)", throughput2intensity(throughputmax), throughputmax);
		CUDA_SAFE_CALL(hipSetDevice(dev_id));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
#if defined WIN32 && !defined _WIN64
		// 2GB limit for hipMalloc
		if (throughputmax > (min(0x7fffffffULL, props.totalGlobalMem) - 256 * 1024 * 1024) / ((520 + 2 * r * (N + 16 * p)) * sizeof(uint32_t)))
#else
		if (throughputmax > (props.totalGlobalMem - 256 * 1024 * 1024) / ((520 + 2 * r * (N + 16 * p)) * sizeof(uint32_t)))
#endif
		{
			applog(LOG_ERR, "Memory Error, mem %d",props.totalGlobalMem);
			mining_has_stopped[thr_id] = true;
			hipStreamDestroy(gpustream[thr_id]);
			proper_exit(2);
		}
		
		size_t hash1_sz = 2 * 16 * r * p * sizeof(uint32_t);	// B
		size_t hash2_sz = 512 * sizeof(uint32_t);				// S(4way)
		size_t hash3_sz = 2 * N * r * sizeof(uint32_t);			// V(16way)
		size_t hash4_sz = 8 * sizeof(uint32_t);					// sha256
		CUDA_SAFE_CALL(hipMalloc(&d_hash1, hash1_sz * throughputmax));
		CUDA_SAFE_CALL(hipMalloc(&d_hash2, hash2_sz * throughputmax));
		CUDA_SAFE_CALL(hipMalloc(&d_hash3, hash3_sz * throughputmax));
		CUDA_SAFE_CALL(hipMalloc(&d_hash4, hash4_sz * throughputmax));

		yescrypt_cpu_init(thr_id, throughputmax, d_hash1, d_hash2, d_hash3, d_hash4);
		mining_has_stopped[thr_id] = false;

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k = 0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	yescrypt_setTarget(thr_id, pdata, key, key_len);

	int warn = 0;
	do {
		uint32_t foundNonce[2] = { 0, 0 };

		yescrypt_cpu_hash_32(thr_id, throughput, pdata[19], foundNonce, ptarget[7], N, r, p);

		if (abort_flag)
		{
			mining_has_stopped[thr_id] = true; 
			hipStreamDestroy(gpustream[thr_id]); 
			pthread_exit(NULL);
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->valid_nonces = 0;
        for(int n=0 ; n<2 ; n++){
            if (foundNonce[n] != 0)
            {
                const uint32_t Htarg = ptarget[7];
                uint32_t _ALIGN(64) vhash[8];
                work->nonces[work->valid_nonces] = foundNonce[n];
                be32enc(&endiandata[19], foundNonce[n]);
                yescrypt_hash_base(vhash, endiandata, N, r, p, key, key_len);
    
                if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
                    bn_set_target_ratio(work, vhash, work->valid_nonces);
                    if(work->nonces[work->valid_nonces] + 1 > pdata[19]){
                        pdata[19] = work->nonces[work->valid_nonces] + 1; 
                    }
                    work->valid_nonces++;
                }
                else if (vhash[7] > Htarg) {
                    gpu_increment_reject(thr_id);
                    if (!warn) {
                        warn++;
                        pdata[19] = work->nonces[work->valid_nonces] + 1;
                        continue;
                    } else {
                        if (!opt_quiet)
                            gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU! ( %08x > %08x )", foundNonce[n],vhash[7],Htarg);
                        warn = 0;
                    }
                }
            }
        }

        if(work->valid_nonces > 0){
            if (work->valid_nonces > 1)
			    applog(LOG_WARNING, "Found multiple nonces : %d, from GPU #%d (%s)", work->valid_nonces, thr_id, device_name[dev_id]);
            return work->valid_nonces;
        }

        if ((uint64_t)throughput + pdata[19] >= max_nonce) {
            pdata[19] = max_nonce;
            break;
        }

        pdata[19] += throughput;

    } while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

    *hashes_done = pdata[19] - first_nonce;
    return 0;
}

int scanhash_yescrypt(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	if (yescrypt_param_N == 0) yescrypt_param_N = 2048;
	if (yescrypt_param_r == 0) yescrypt_param_r = 8;
	if (yescrypt_param_p == 0) yescrypt_param_p = 1;
	return  scanhash_yescrypt_base(thr_id, work, max_nonce, hashes_done, yescrypt_param_N, yescrypt_param_r, yescrypt_param_p, yescrypt_key, yescrypt_key_len);
}

int scanhash_yescryptr8(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	return  scanhash_yescrypt_base(thr_id, work, max_nonce, hashes_done, 2048, 8, 1, "Client Key", 10);
}

int scanhash_yescryptr16(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	return  scanhash_yescrypt_base(thr_id, work, max_nonce, hashes_done, 4096, 16, 1, "Client Key", 10);
}

int scanhash_yescryptr16v2(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	return  scanhash_yescrypt_base(thr_id, work, max_nonce, hashes_done, 4096, 16, 4, "PPTPPubKey", 10);
}

int scanhash_yescryptr32(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	return  scanhash_yescrypt_base(thr_id, work, max_nonce, hashes_done, 4096, 32, 1, "WaviBanana", 10);
}

extern "C" void free_yescrypt(int thr_id)
{
    if (!init[thr_id])
        return;

    hipDeviceSynchronize();

    yescrypt_cpu_free(thr_id);

    hipDeviceSynchronize();
    init[thr_id] = false;
}
